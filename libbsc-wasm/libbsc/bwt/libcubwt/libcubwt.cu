#include "hip/hip_runtime.h"
/*--

This file is a part of libcubwt, a library for CUDA accelerated
burrows wheeler transform construction.

   Copyright (c) 2022-2023 Ilya Grebnov <ilya.grebnov@gmail.com>

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.

Please see the file LICENSE for full copyright and license details.

--*/

#include "libcubwt.cuh"

#if defined(_MSC_VER) && defined(__INTELLISENSE__)
    #define __launch_bounds__(block_size) /* */
    #define __HIPCC__

    #include <vector_functions.h>
    #include <hip/device_functions.h>
    #include <>
#endif

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include <utility>

#if defined(__GNUC__) || defined(__clang__) || defined(__HIPCC__)
    #define RESTRICT __restrict__
#elif defined(_MSC_VER) || defined(__INTEL_COMPILER)
    #define RESTRICT __restrict
#else
    #define RESTRICT /* */
#endif

#ifndef __CUDA_ARCH__
    #define CUDA_DEVICE_ARCH                0
#else
    #define CUDA_DEVICE_ARCH                __CUDA_ARCH__
#endif

#if CUDA_DEVICE_ARCH == 750
    #define CUDA_SM_THREADS                 (1024)
#elif CUDA_DEVICE_ARCH == 860 || CUDA_DEVICE_ARCH == 870 || CUDA_DEVICE_ARCH == 890
    #define CUDA_SM_THREADS                 (1536)
#else
    #define CUDA_SM_THREADS                 (2048)
#endif

#if CUDA_DEVICE_ARCH == 860 || CUDA_DEVICE_ARCH == 870 || CUDA_DEVICE_ARCH == 890
    #define CUDA_BLOCK_THREADS              (768)
#else
    #define CUDA_BLOCK_THREADS              (512)
#endif

#define CUDA_WARP_THREADS                   (32)
#define CUDA_DEVICE_PADDING                 (12 * 768)

typedef struct LIBCUBWT_DEVICE_STORAGE
{
    void *          device_rsort_temp_storage;
    size_t          device_rsort_temp_storage_size;

    void *          device_ssort_temp_storage;
    size_t          device_ssort_temp_storage_size;

    uint8_t *       device_T;
    uint8_t *       device_heads;

    uint32_t *      device_SA;
    uint32_t *      device_ISA;

    uint32_t *      device_keys;
    uint32_t *      device_offsets;

    uint32_t *      device_temp_SA;
    uint32_t *      device_temp_ISA;
    uint32_t *      device_temp_keys;

    uint64_t *      device_SA_temp_SA;
    uint64_t *      device_keys_temp_keys;
    uint64_t *      device_offsets_ISA;

    uint4 *         device_descriptors_large;
    uint4 *         device_descriptors_copy;
    uint2 *         device_descriptors_small;

    void *          device_storage;
    int32_t         device_L2_cache_bits;

    void *          host_pinned_storage;
    size_t          host_pinned_storage_size;

    int64_t         max_length;
    uint32_t        num_unsorted_segments;
    uint32_t        num_unsorted_suffixes;
    
    uint32_t        cuda_block_threads;
    hipStream_t    hip_stream;
} LIBCUBWT_DEVICE_STORAGE;

static int64_t libcubwt_get_error_code(hipError_t status)
{
    return
        status == hipErrorOutOfMemory     ? LIBCUBWT_GPU_NOT_ENOUGH_MEMORY :
        status == cudaErrorDevicesUnavailable   ? LIBCUBWT_GPU_NOT_SUPPORTED :
        status == hipErrorNoDevice             ? LIBCUBWT_GPU_NOT_SUPPORTED :
        LIBCUBWT_GPU_ERROR;
}

static hipError_t libcubwt_cuda_safe_call(const char * filename, int32_t line, hipError_t result, hipError_t status = hipSuccess)
{
#if !defined(NDEBUG)
    if (result != hipSuccess)
    {
        fprintf(stderr, "%s(%d): libcubwt_cuda_safe_call failed %d: '%s'.\n", filename, line, result, hipGetErrorString(result));
        fflush(stderr);
    }
#else
    (void)(filename); (void)(line);
#endif

    return result != hipSuccess ? result : status;
}

template <typename T>
__device__ __forceinline__ T libcubwt_warp_reduce_sum(T value) 
{
#if CUDA_DEVICE_ARCH >= 800 && !defined(__CUDA__)
    return __reduce_add_sync((uint32_t)-1, value);
#else

    #pragma unroll
    for (uint32_t mask = CUDA_WARP_THREADS / 2; mask > 0; mask >>= 1)
    {
        value = hipcub::Sum()(value, __shfl_xor_sync((uint32_t)-1, value, mask, CUDA_WARP_THREADS));
    }

    return value;
#endif
}

template <typename T>
__device__ __forceinline__ T libcubwt_warp_reduce_max(T value) 
{
#if CUDA_DEVICE_ARCH >= 800 && !defined(__CUDA__)
    return __reduce_max_sync((uint32_t)-1, value);
#else

    #pragma unroll
    for (uint32_t mask = CUDA_WARP_THREADS / 2; mask > 0; mask >>= 1)
    {
        value = hipcub::Max()(value, __shfl_xor_sync((uint32_t)-1, value, mask, CUDA_WARP_THREADS));
    }

    return value;
#endif
}

template <typename T>
__device__ __forceinline__ void libcubwt_delay_or_prevent_hoisting(T delay)
{
#if CUDA_DEVICE_ARCH >= 700
    __nanosleep(delay);
#else
    __threadfence_block(); (void)(delay);
#endif
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_gather_values_uint32_kernel(const uint32_t * device_idx, const uint32_t * RESTRICT device_src, uint32_t * device_dst, uint32_t m)
{
    const uint32_t block_index = blockIdx.x * CUDA_BLOCK_THREADS * 4;

    device_idx += block_index; device_dst += block_index; m -= block_index;

    if (m >= CUDA_BLOCK_THREADS * 4)
    {
        const uint4 indexes = *(uint4 *)(device_idx + threadIdx.x * 4);

        *(uint4 *)(device_dst + threadIdx.x * 4) = make_uint4(
            __ldg(device_src + indexes.x),
            __ldg(device_src + indexes.y),
            __ldg(device_src + indexes.z),
            __ldg(device_src + indexes.w));
    }
    else
    {
        for (uint32_t thread_index = threadIdx.x; thread_index < m; thread_index += CUDA_BLOCK_THREADS)
        {
            device_dst[thread_index] = __ldg(device_src + device_idx[thread_index]);
        }
    }
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_scatter_values_uint32_kernel(const uint32_t * RESTRICT device_idx, const uint32_t * RESTRICT device_src, uint32_t * RESTRICT device_dst, uint32_t m)
{
    const uint32_t block_index = blockIdx.x * CUDA_BLOCK_THREADS * 4;

    device_idx += block_index; device_src += block_index; m -= block_index;

    if (m >= CUDA_BLOCK_THREADS * 4)
    {
        const uint4 indexes = __ldg((uint4 *)(device_idx + threadIdx.x * 4));
        const uint4 values  = __ldg((uint4 *)(device_src + threadIdx.x * 4));

        device_dst[indexes.x] = values.x;
        device_dst[indexes.y] = values.y;
        device_dst[indexes.z] = values.z;
        device_dst[indexes.w] = values.w;
    }
    else
    {
        for (uint32_t thread_index = threadIdx.x; thread_index < m; thread_index += CUDA_BLOCK_THREADS)
        {
            device_dst[__ldg(device_idx + thread_index)] = __ldg(device_src + thread_index);
        }
    }
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_permute_block_values_uint32_kernel(const uint32_t * RESTRICT device_idx, const uint32_t * RESTRICT device_src, uint32_t * RESTRICT device_dst, uint32_t n)
{
    __shared__ __align__(32) uint32_t cache[16 * CUDA_BLOCK_THREADS];

    const uint32_t block_index = blockIdx.x * CUDA_BLOCK_THREADS * 16;

    device_idx += block_index; device_src += block_index; device_dst += block_index; n -= block_index;

    if (n >= CUDA_BLOCK_THREADS * 16)
    {
        {
            const uint32_t * RESTRICT thread_idx   = device_idx + threadIdx.x * 4;
            const uint32_t * RESTRICT thread_src   = device_src + threadIdx.x * 4;
                  uint32_t * RESTRICT thread_cache = cache - block_index;

            #pragma unroll
            for (uint32_t round = 0; round < 4; round += 1)
            {
                const uint4 indexes = __ldg((uint4 *)(thread_idx));
                const uint4 values  = __ldg((uint4 *)(thread_src));

                thread_cache[indexes.x] = values.x;
                thread_cache[indexes.y] = values.y;
                thread_cache[indexes.z] = values.z;
                thread_cache[indexes.w] = values.w;

                thread_idx += 4 * CUDA_BLOCK_THREADS; thread_src += 4 * CUDA_BLOCK_THREADS;
            }
        }

        __syncthreads();

        {
            const uint32_t * RESTRICT thread_cache = cache      + threadIdx.x * 4;
                  uint32_t * RESTRICT thread_dst   = device_dst + threadIdx.x * 4;

            #pragma unroll
            for (uint32_t round = 0; round < 4; round += 1)
            {
                *(uint4 *)(thread_dst) = *(uint4 *)(thread_cache);

                thread_cache += 4 * CUDA_BLOCK_THREADS; thread_dst += 4 * CUDA_BLOCK_THREADS;
            }
        }
    }
    else
    {
        {
            uint32_t * RESTRICT thread_cache = cache - block_index;

            for (uint32_t thread_index = threadIdx.x; thread_index < n; thread_index += CUDA_BLOCK_THREADS)
            {
                thread_cache[__ldg(device_idx + thread_index)] = __ldg(device_src + thread_index);
            }
        }

        __syncthreads();

        {
            for (uint32_t thread_index = threadIdx.x; thread_index < n; thread_index += CUDA_BLOCK_THREADS)
            {
                device_dst[thread_index] = cache[thread_index];
            }
        }
    }
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_scatter_values_uint64_kernel(const uint32_t * RESTRICT device_idx, const uint64_t * RESTRICT device_src, uint64_t * RESTRICT device_dst, uint32_t m)
{
    const uint32_t block_index = blockIdx.x * CUDA_BLOCK_THREADS * 2;

    device_idx += block_index; device_src += block_index; m -= block_index;

    if (m >= CUDA_BLOCK_THREADS * 2)
    {
        const uint2      indexes = __ldg((uint2      *)(device_idx + threadIdx.x * 2));
        const ulonglong2 values  = __ldg((ulonglong2 *)(device_src + threadIdx.x * 2));

        device_dst[indexes.x] = values.x;
        device_dst[indexes.y] = values.y;
    }
    else
    {
        for (uint32_t thread_index = threadIdx.x; thread_index < m; thread_index += CUDA_BLOCK_THREADS)
        {
            device_dst[__ldg(device_idx + thread_index)] = __ldg(device_src + thread_index);
        }
    }
}

static hipError_t libcubwt_gather_scatter_values_uint32(LIBCUBWT_DEVICE_STORAGE * storage, uint32_t * device_src_idx, uint32_t * device_src, uint32_t * device_dst_idx, uint32_t * device_dst, int64_t m, int64_t n, uint32_t * device_temp1, uint32_t * device_temp2)
{
    hipError_t status = hipSuccess;

    hipcub::DoubleBuffer<uint32_t> db_src_index_value(device_src_idx, device_temp1);
    hipcub::DoubleBuffer<uint32_t> db_dst_index(device_dst_idx, device_temp2);

    int32_t sort_end_bit        = 0; while ((n - 1) >= ((int64_t)1 << sort_end_bit)) { sort_end_bit += 1; }
    int32_t sort_aligned_bits   = (sort_end_bit > storage->device_L2_cache_bits - 2) ? (sort_end_bit - storage->device_L2_cache_bits + 2 + 7) & (-8) : 0;
    int32_t sort_start_bit      = std::max(0, sort_end_bit - sort_aligned_bits);

    if (sort_start_bit < sort_end_bit)
    {
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
            storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
            db_src_index_value, db_dst_index,
            (uint32_t)m,
            sort_start_bit, sort_end_bit,
            storage->hip_stream));
    }

    if (status == hipSuccess)
    {
        int64_t n_gather_scatter_blocks = (m + storage->cuda_block_threads * 4 - 1) / (storage->cuda_block_threads * 4);

        libcubwt_gather_values_uint32_kernel<<<(uint32_t)n_gather_scatter_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(db_src_index_value.Current(), device_src, db_src_index_value.Current(), (uint32_t)m);

        if (sort_start_bit < sort_end_bit)
        {
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
                storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
                db_dst_index, db_src_index_value,
                (uint32_t)m,
                sort_start_bit, sort_end_bit,
                storage->hip_stream));
        }

        if (status == hipSuccess)
        {
            libcubwt_scatter_values_uint32_kernel<<<(uint32_t)n_gather_scatter_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(db_dst_index.Current(), db_src_index_value.Current(), device_dst, (uint32_t)m);
        }
    }

    return status;
}

static hipError_t libcubwt_scatter_values_uint32(LIBCUBWT_DEVICE_STORAGE * storage, uint32_t * device_idx, uint32_t * device_src, uint32_t * device_dst, int64_t m, int64_t n, uint32_t * device_temp1, uint32_t * device_temp2)
{
    hipError_t status = hipSuccess;

    hipcub::DoubleBuffer<uint32_t> db_index(device_idx, device_temp1);
    hipcub::DoubleBuffer<uint32_t> db_value(device_src, device_temp2);

    int32_t sort_end_bit        = 0; while ((n - 1) >= ((int64_t)1 << sort_end_bit)) { sort_end_bit += 1; }
    int32_t sort_aligned_bits   = (sort_end_bit > storage->device_L2_cache_bits - 2) ? (sort_end_bit - storage->device_L2_cache_bits + 2 + 7) & (-8) : 0;
    int32_t sort_start_bit      = std::max(0, sort_end_bit - sort_aligned_bits);

    if (sort_start_bit < sort_end_bit)
    {
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
            storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
            db_index, db_value,
            (uint32_t)m,
            sort_start_bit, sort_end_bit,
            storage->hip_stream));
    }

    if (status == hipSuccess)
    {
        int64_t n_scatter_blocks = (m + storage->cuda_block_threads * 4 - 1) / (storage->cuda_block_threads * 4);

        libcubwt_scatter_values_uint32_kernel<<<(uint32_t)n_scatter_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(db_index.Current(), db_value.Current(), device_dst, (uint32_t)m);
    }

    return status;
}

static hipError_t libcubwt_permute_values_uint32(LIBCUBWT_DEVICE_STORAGE * storage, uint32_t * device_idx, uint32_t * device_src, uint32_t * device_dst, int64_t n, uint32_t * device_temp1, uint32_t * device_temp2)
{
    hipError_t status = hipSuccess;

    hipcub::DoubleBuffer<uint32_t> db_index(device_idx, device_temp1);
    hipcub::DoubleBuffer<uint32_t> db_value(device_src, device_temp2);

    int32_t sort_end_bit        = 0; while ((n - 1) >= ((int64_t)1 << sort_end_bit)) { sort_end_bit += 1; }
    int32_t sort_aligned_bits   = (sort_end_bit > storage->device_L2_cache_bits - 2) ? (sort_end_bit - storage->device_L2_cache_bits + 2 + 7) & (-8) : 0;
    int32_t sort_start_bit      = std::max(0, sort_end_bit - sort_aligned_bits);

    if (sort_start_bit < sort_end_bit)
    {
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
            storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
            db_index, db_value,
            (uint32_t)n,
            sort_start_bit, sort_end_bit,
            storage->hip_stream));
    }

    if (status == hipSuccess)
    {
        if (((storage->cuda_block_threads * 16) % ((int64_t)1 << sort_start_bit)) == 0)
        {
            int64_t n_permute_blocks = (n + storage->cuda_block_threads * 16 - 1) / (storage->cuda_block_threads * 16);

            libcubwt_permute_block_values_uint32_kernel<<<(uint32_t)n_permute_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(db_index.Current(), db_value.Current(), device_dst, (uint32_t)n);
        }
        else
        {
            int64_t n_scatter_blocks = (n + storage->cuda_block_threads * 4 - 1) / (storage->cuda_block_threads * 4);

            libcubwt_scatter_values_uint32_kernel<<<(uint32_t)n_scatter_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(db_index.Current(), db_value.Current(), device_dst, (uint32_t)n);
        }
    }

    return status;
}

static hipError_t libcubwt_scatter_values_uint64(LIBCUBWT_DEVICE_STORAGE * storage, hipcub::DoubleBuffer<uint32_t> & db_index, hipcub::DoubleBuffer<uint64_t> & db_value, int64_t m, int64_t n, int64_t k = 0)
{
    hipError_t status = hipSuccess;

    int32_t sort_end_bit        = 0; while ((n - 1) >= ((int64_t)1 << sort_end_bit)) { sort_end_bit += 1; }
    int32_t sort_aligned_bits   = (sort_end_bit > storage->device_L2_cache_bits - 3) ? (sort_end_bit - storage->device_L2_cache_bits + 3 + 7) & (-8) : 0;
    int32_t sort_start_bit      = std::max(0, sort_end_bit - sort_aligned_bits);

    if (sort_start_bit < sort_end_bit)
    {
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
            storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
            db_index, db_value,
            (uint32_t)m,
            sort_start_bit, sort_end_bit,
            storage->hip_stream));
    }

    if (status == hipSuccess)
    {
        int64_t n_scatter_blocks = (m + storage->cuda_block_threads * 2 - 1) / (storage->cuda_block_threads * 2);

        libcubwt_scatter_values_uint64_kernel<<<(uint32_t)n_scatter_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(db_index.Current(), db_value.Current(), db_value.Alternate() - k, (uint32_t)m);

        db_index.selector ^= 1;
        db_value.selector ^= 1;
    }

    return status;
}

template <bool extra_sentinel_bits>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_initialize_device_arrays_kernel(const uint8_t * RESTRICT device_T, uint32_t * RESTRICT device_SA, uint64_t * RESTRICT device_keys)
{
    __shared__ __align__(32) uint4 prefixes[4 * CUDA_BLOCK_THREADS];

    {
        device_T += blockIdx.x * CUDA_BLOCK_THREADS * 12 + threadIdx.x * 16;
        if (threadIdx.x < (12 * CUDA_BLOCK_THREADS + 8 + 15) / 16) { prefixes[threadIdx.x] = __ldg((uint4 *)device_T); }

        __syncthreads();
    }

    {
        uint32_t * RESTRICT thread_cache    = ((uint32_t *)prefixes) + threadIdx.x * 3;
        uint4 *    RESTRICT thread_prefixes = ((uint4 *   )prefixes) + threadIdx.x * 4;

        const uint32_t b0 = thread_cache[0];
        const uint32_t b1 = thread_cache[1];
        const uint32_t b2 = thread_cache[2];
        const uint32_t b3 = thread_cache[3];
        const uint32_t b4 = thread_cache[4];

        __syncthreads();

        thread_prefixes[0] = make_uint4
        (
            __byte_perm(b1, b2, 0x1234) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(b0, b1, 0x1234),
            __byte_perm(b1, b2, 0x2345) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(b0, b1, 0x2345)
        );

        thread_prefixes[1] = make_uint4
        (
            __byte_perm(b2, b3, 0x0123) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(b1, b2, 0x0123),
            __byte_perm(b2, b3, 0x1234) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(b1, b2, 0x1234)
        );

        thread_prefixes[2] = make_uint4
        (
            __byte_perm(b2, b3, 0x3456) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(b1, b2, 0x3456),
            __byte_perm(b3, b4, 0x0123) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(b2, b3, 0x0123)
        );

        thread_prefixes[3] = make_uint4
        (
            __byte_perm(b3, b4, 0x2345) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(b2, b3, 0x2345),
            __byte_perm(b3, b4, 0x3456) | (extra_sentinel_bits ? (uint32_t)7 : (uint32_t)1), __byte_perm(b2, b3, 0x3456)
        );

        __syncwarp();
    }

    {
        const uint32_t block_index = blockIdx.x * CUDA_BLOCK_THREADS * 8;

        {
            uint32_t thread_index = block_index + threadIdx.x * 4; device_SA += thread_index;
            ((uint4 *)device_SA)[0] = make_uint4(thread_index + 0, thread_index + 1, thread_index + 2, thread_index + 3);

            thread_index += CUDA_BLOCK_THREADS * 4; device_SA += CUDA_BLOCK_THREADS * 4;
            ((uint4 *)device_SA)[0] = make_uint4(thread_index + 0, thread_index + 1, thread_index + 2, thread_index + 3);
        }

        {
            device_keys += block_index;

            uint4 * RESTRICT thread_prefixes = (uint4 *)prefixes    + ((threadIdx.x / CUDA_WARP_THREADS) * CUDA_WARP_THREADS * 4) + (threadIdx.x % CUDA_WARP_THREADS);
            uint4 * RESTRICT thread_keys     = (uint4 *)device_keys + ((threadIdx.x / CUDA_WARP_THREADS) * CUDA_WARP_THREADS * 4) + (threadIdx.x % CUDA_WARP_THREADS);

            thread_keys[0] = thread_prefixes[0]; thread_keys += CUDA_WARP_THREADS; thread_prefixes += CUDA_WARP_THREADS;
            thread_keys[0] = thread_prefixes[0]; thread_keys += CUDA_WARP_THREADS; thread_prefixes += CUDA_WARP_THREADS;
            thread_keys[0] = thread_prefixes[0]; thread_keys += CUDA_WARP_THREADS; thread_prefixes += CUDA_WARP_THREADS;
            thread_keys[0] = thread_prefixes[0];
        }
    }
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, 1)
static void libcubwt_set_sentinel_values_kernel(uint8_t * RESTRICT device_T_end, uint64_t * RESTRICT device_keys_end, uint64_t k0, uint64_t k1, uint64_t k2, uint64_t k3, uint64_t k4, uint64_t k5, uint64_t k6, uint64_t k7)
{
    device_T_end[0] = 0;
    device_T_end[1] = 0;
    device_T_end[2] = 0;

    device_keys_end[-8] = k0;
    device_keys_end[-7] = k1;
    device_keys_end[-6] = k2;
    device_keys_end[-5] = k3;
    device_keys_end[-4] = k4;
    device_keys_end[-3] = k5;
    device_keys_end[-2] = k6;
    device_keys_end[-1] = k7;
}

static hipError_t libcubwt_initialize_device_arrays(LIBCUBWT_DEVICE_STORAGE * storage, const uint8_t * T, int64_t reduced_n, int64_t expanded_n, int64_t input_n)
{
    hipError_t status = hipSuccess;

    if ((status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->device_T, T, (size_t)input_n, hipMemcpyHostToDevice, storage->hip_stream))) == hipSuccess)
    {
        int64_t n_initialize_blocks = 1 + (expanded_n / (storage->cuda_block_threads * 12));

        bool extra_sentinel_bits = (expanded_n - input_n >= 2) || (T[input_n - 1] == 0);
        if (extra_sentinel_bits)
        {
            libcubwt_initialize_device_arrays_kernel<true><<<(uint32_t)n_initialize_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(storage->device_T, storage->device_SA, storage->device_keys_temp_keys);
        }
        else
        {
            libcubwt_initialize_device_arrays_kernel<false><<<(uint32_t)n_initialize_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(storage->device_T, storage->device_SA, storage->device_keys_temp_keys);
        }

        {
            uint64_t c0 = (expanded_n - 11 < input_n) ? T[expanded_n - 11] : (uint64_t)0;
            uint64_t c1 = (expanded_n - 10 < input_n) ? T[expanded_n - 10] : (uint64_t)0;
            uint64_t c2 = (expanded_n -  9 < input_n) ? T[expanded_n -  9] : (uint64_t)0;
            uint64_t c3 = (expanded_n -  8 < input_n) ? T[expanded_n -  8] : (uint64_t)0;
            uint64_t c4 = (expanded_n -  7 < input_n) ? T[expanded_n -  7] : (uint64_t)0;
            uint64_t c5 = (expanded_n -  6 < input_n) ? T[expanded_n -  6] : (uint64_t)0;
            uint64_t c6 = (expanded_n -  5 < input_n) ? T[expanded_n -  5] : (uint64_t)0;
            uint64_t c7 = (expanded_n -  4 < input_n) ? T[expanded_n -  4] : (uint64_t)0;
            uint64_t c8 = (expanded_n -  3 < input_n) ? T[expanded_n -  3] : (uint64_t)0;
            uint64_t c9 = (expanded_n -  2 < input_n) ? T[expanded_n -  2] : (uint64_t)0;
            uint64_t ca = (expanded_n -  1 < input_n) ? T[expanded_n -  1] : (uint64_t)0;

            uint64_t k0 = (c0 << 56) | (c1 << 48) | (c2 << 40) | (c3 << 32) | (c4 << 24) | (c5 << 16) | (c6 << 8) | (c7 << 0) | (extra_sentinel_bits ? 7 : 1);
            uint64_t k1 = (c1 << 56) | (c2 << 48) | (c3 << 40) | (c4 << 32) | (c5 << 24) | (c6 << 16) | (c7 << 8) | (c8 << 0) | (extra_sentinel_bits ? 7 : 1);

            uint64_t k2 = (c3 << 56) | (c4 << 48) | (c5 << 40) | (c6 << 32) | (c7 << 24) | (c8 << 16) | (c9 << 8) | (ca << 0) | (extra_sentinel_bits ? 7 : 0);
            uint64_t k3 = (c4 << 56) | (c5 << 48) | (c6 << 40) | (c7 << 32) | (c8 << 24) | (c9 << 16) | (ca << 8) | (extra_sentinel_bits ? 6 : 0);
            
            uint64_t k4 = (c6 << 56) | (c7 << 48) | (c8 << 40) | (c9 << 32) | (ca << 24) | (extra_sentinel_bits ? 4 : 0);
            uint64_t k5 = (c7 << 56) | (c8 << 48) | (c9 << 40) | (ca << 32) | (extra_sentinel_bits ? 3 : 0);

            uint64_t k6 = (c9 << 56) | (ca << 48) | (extra_sentinel_bits ? 1 : 0);
            uint64_t k7 = (ca << 56);

            libcubwt_set_sentinel_values_kernel<<<1, 1, 0, storage->hip_stream>>>(storage->device_T + input_n, storage->device_keys_temp_keys + reduced_n, k0, k1, k2, k3, k4, k5, k6, k7);
        }

        storage->num_unsorted_segments      = (uint32_t)1;
        storage->num_unsorted_suffixes      = (uint32_t)reduced_n;
    }

    return status;
}

static hipError_t libcubwt_sort_suffixes_by_prefix(LIBCUBWT_DEVICE_STORAGE * storage, int64_t n)
{
    hipcub::DoubleBuffer<uint64_t> db_keys(storage->device_keys_temp_keys, storage->device_offsets_ISA);
    hipcub::DoubleBuffer<uint32_t> db_SA(storage->device_SA, storage->device_temp_SA);

    hipError_t status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
        storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
        db_keys, db_SA,
        (uint32_t)n,
        0, 64,
        storage->hip_stream));

    if (db_keys.selector) 
    { 
        std::swap(storage->device_keys_temp_keys, storage->device_offsets_ISA);

        std::swap(storage->device_keys, storage->device_offsets); 
        std::swap(storage->device_temp_keys, storage->device_ISA);
    }

    if (db_SA.selector)
    {
        std::swap(storage->device_SA, storage->device_temp_SA);
    }

    return status;
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_rank_and_segment_suffixes_initialization_kernel(uint32_t * RESTRICT device_SA, uint64_t * RESTRICT device_keys, uint8_t * RESTRICT device_heads, uint4 * RESTRICT device_descriptors_large, uint2 * RESTRICT device_descriptors_small, uint32_t n)
{
    const uint32_t thread_index = blockIdx.x * CUDA_BLOCK_THREADS + threadIdx.x;

    device_descriptors_large += thread_index;
    device_descriptors_small += thread_index;

    device_descriptors_large[0] = make_uint4(0, 0, 0, 0);
    device_descriptors_small[0] = make_uint2(0, 0);

    if (blockIdx.x == 0)
    {
        if (threadIdx.x < CUDA_WARP_THREADS)
        {
            device_descriptors_large[-CUDA_WARP_THREADS] = make_uint4((uint32_t)-1, 0, 0, 0);
            device_descriptors_small[-CUDA_WARP_THREADS] = make_uint2((uint32_t)-1, 0);
        }

        {
            uint64_t key = (threadIdx.x % 2 == 0) ? 0 : (uint64_t)-1;

            device_SA += threadIdx.x; device_keys += threadIdx.x; device_heads += threadIdx.x;

            if (threadIdx.x < 2)
            {
                device_keys [-2] = key;
                device_heads[-2] = 1;
            }

            device_SA += n; device_keys += n; device_heads += n;

            device_SA   [0 * CUDA_BLOCK_THREADS] = n + threadIdx.x + 0 * CUDA_BLOCK_THREADS;
            device_SA   [1 * CUDA_BLOCK_THREADS] = n + threadIdx.x + 1 * CUDA_BLOCK_THREADS;
            device_SA   [2 * CUDA_BLOCK_THREADS] = n + threadIdx.x + 2 * CUDA_BLOCK_THREADS;
            device_SA   [3 * CUDA_BLOCK_THREADS] = n + threadIdx.x + 3 * CUDA_BLOCK_THREADS;

            device_keys [0 * CUDA_BLOCK_THREADS] = key;
            device_keys [1 * CUDA_BLOCK_THREADS] = key;
            device_keys [2 * CUDA_BLOCK_THREADS] = key;
            device_keys [3 * CUDA_BLOCK_THREADS] = key;

            device_heads[0 * CUDA_BLOCK_THREADS] = 1;
            device_heads[1 * CUDA_BLOCK_THREADS] = 1;
            device_heads[2 * CUDA_BLOCK_THREADS] = 1;
            device_heads[3 * CUDA_BLOCK_THREADS] = 1;
        }
    }
}

template <bool scatter_ranks_directly>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_rank_and_segment_suffixes_initiatory_kernel(
    const uint32_t *    RESTRICT device_SA,
    const uint64_t *    RESTRICT device_keys,
    uint8_t *           RESTRICT device_heads,
    uint32_t *          RESTRICT device_ISA,
    uint32_t *          RESTRICT device_offsets_begin,
    uint32_t *          RESTRICT device_offsets_end,
    uint4 *             RESTRICT device_descriptors
)
{
    __shared__ __align__(32) uint2 warp_state[1 + CUDA_WARP_THREADS];

    uint32_t    thread_exclusive_suffix_rank;
    uint32_t    thread_suffix_rank[4];

    uint32_t    thread_exclusive_segment_index;
    uint32_t    thread_segment_index[4];

    {
        __shared__ __align__(32) ulonglong2 cache[1 + 2 * CUDA_BLOCK_THREADS];

        {
            device_keys += blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 2;

            if (threadIdx.x == 0) { cache[0] = __ldg((ulonglong2 *)(device_keys - 2)); }
            cache[1 + threadIdx.x + 0 * CUDA_BLOCK_THREADS] = __ldg((ulonglong2 *)(device_keys + 0 * CUDA_BLOCK_THREADS));
            cache[1 + threadIdx.x + 1 * CUDA_BLOCK_THREADS] = __ldg((ulonglong2 *)(device_keys + 2 * CUDA_BLOCK_THREADS));
        }

        __syncthreads();

        {
            const uint32_t block_index  = blockIdx.x * CUDA_BLOCK_THREADS * 4;
            const uint32_t thread_index = block_index + threadIdx.x * 4;

            ulonglong2 key_a = cache[2 * threadIdx.x + 0];
            ulonglong2 key_b = cache[2 * threadIdx.x + 1];
            ulonglong2 key_c = cache[2 * threadIdx.x + 2];

            uchar4 thread_new_heads = make_uchar4(
                (key_a.y != key_b.x) ? (uint8_t)1 : (uint8_t)0,
                (key_b.x != key_b.y) ? (uint8_t)1 : (uint8_t)0,
                (key_b.y != key_c.x) ? (uint8_t)1 : (uint8_t)0,
                (key_c.x != key_c.y) ? (uint8_t)1 : (uint8_t)0);

            *(uchar4 *)(device_heads + thread_index) = thread_new_heads;

            thread_suffix_rank[0] = (thread_new_heads.x != 0) ? (thread_index + 0) : 0;
            thread_suffix_rank[1] = (thread_new_heads.y != 0) ? (thread_index + 1) : thread_suffix_rank[0];
            thread_suffix_rank[2] = (thread_new_heads.z != 0) ? (thread_index + 2) : thread_suffix_rank[1];
            thread_suffix_rank[3] = (thread_new_heads.w != 0) ? (thread_index + 3) : thread_suffix_rank[2];

            thread_segment_index[0] = ((thread_new_heads.x != 0) && (key_a.x == key_a.y));
            thread_segment_index[1] = thread_segment_index[0] + ((thread_new_heads.y != 0) && (thread_new_heads.x == 0));
            thread_segment_index[2] = thread_segment_index[1] + ((thread_new_heads.z != 0) && (thread_new_heads.y == 0));
            thread_segment_index[3] = thread_segment_index[2] + ((thread_new_heads.w != 0) && (thread_new_heads.z == 0));
        }
    }

    {
        uint32_t thread_inclusive_suffix_rank;
        uint32_t thread_inclusive_segment_index;

        typedef hipcub::WarpScan<uint32_t> WarpScan;

        __shared__ typename WarpScan::TempStorage warp_scan_storage[CUDA_WARP_THREADS];

        WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_suffix_rank[3]  , thread_inclusive_suffix_rank  , thread_exclusive_suffix_rank  , (uint32_t)0, hipcub::Max());
        WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_segment_index[3], thread_inclusive_segment_index, thread_exclusive_segment_index, (uint32_t)0, hipcub::Sum());

        if ((threadIdx.x % CUDA_WARP_THREADS) == (CUDA_WARP_THREADS - 1))
        {
            warp_state[threadIdx.x / CUDA_WARP_THREADS] = make_uint2(thread_inclusive_suffix_rank, thread_inclusive_segment_index);
        }

        __syncthreads();
    }

    {
        if (threadIdx.x < CUDA_WARP_THREADS)
        {
            uint32_t            block_exclusive_suffix_rank   = 0;
            uint32_t            block_exclusive_segment_index = 0;

            uint32_t            warp_inclusive_suffix_rank;
            uint32_t            warp_inclusive_segment_index;

            {
                typedef hipcub::WarpScan<uint32_t> WarpScan;

                __shared__ typename WarpScan::TempStorage warp_scan_storage;

                uint2 warp_inclusive_state = warp_state[threadIdx.x];

                WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state.x, warp_inclusive_suffix_rank  , hipcub::Max());
                WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state.y, warp_inclusive_segment_index, hipcub::Sum());
            }

            {
                const uint32_t descriptor_status_aggregate_not_ready        = 0;
                const uint32_t descriptor_status_partial_aggregate_ready    = 1;
                const uint32_t descriptor_status_full_aggregate_ready       = 4;

                if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                {
                    hipcub::ThreadStore<hipcub::STORE_CG>(device_descriptors + blockIdx.x, make_uint4(descriptor_status_partial_aggregate_ready, 0, warp_inclusive_suffix_rank, warp_inclusive_segment_index));
                }

                {
                    uint4 * RESTRICT descriptors_lookback = device_descriptors + blockIdx.x + threadIdx.x;

                    int32_t full_aggregate_lane, delay = 8;
                    do
                    {
                        descriptors_lookback -= CUDA_WARP_THREADS;

                        uint4 block_descriptor;
                        do
                        {
                            libcubwt_delay_or_prevent_hoisting(delay <<= 1);

                            block_descriptor = hipcub::ThreadLoad<hipcub::LOAD_CG>(descriptors_lookback);
                        } while (__any_sync((uint32_t)-1, block_descriptor.x == descriptor_status_aggregate_not_ready));

                        delay = 0;

                        {
                            full_aggregate_lane     = 31 - __clz((int32_t)__ballot_sync((uint32_t)-1, block_descriptor.x != descriptor_status_partial_aggregate_ready));
                            block_descriptor.z      = (((int32_t)threadIdx.x) >= full_aggregate_lane) ? block_descriptor.z : 0;
                            block_descriptor.w      = (((int32_t)threadIdx.x) >= full_aggregate_lane) ? block_descriptor.w : 0;
                        }

                        {
                            block_exclusive_suffix_rank      = hipcub::Max()(block_exclusive_suffix_rank  , libcubwt_warp_reduce_max(block_descriptor.z));
                            block_exclusive_segment_index    = hipcub::Sum()(block_exclusive_segment_index, libcubwt_warp_reduce_sum(block_descriptor.w));
                        }

                    } while (full_aggregate_lane == -1);

                    warp_inclusive_suffix_rank      = hipcub::Max()(warp_inclusive_suffix_rank  , block_exclusive_suffix_rank  );
                    warp_inclusive_segment_index    = hipcub::Sum()(warp_inclusive_segment_index, block_exclusive_segment_index);
                }

                if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                {
                    hipcub::ThreadStore<hipcub::STORE_CG>(device_descriptors + blockIdx.x, make_uint4(descriptor_status_full_aggregate_ready, 0, warp_inclusive_suffix_rank, warp_inclusive_segment_index));
                }
            }

            {
                if (threadIdx.x == 0)
                {
                    warp_state[0] = make_uint2(block_exclusive_suffix_rank, block_exclusive_segment_index);
                }

                warp_state[1 + threadIdx.x] = make_uint2(warp_inclusive_suffix_rank, warp_inclusive_segment_index);
            }
        }

        __syncthreads();
    }

    {
        uint2 warp_exclusive_state              = warp_state[threadIdx.x / CUDA_WARP_THREADS];
        
        thread_exclusive_suffix_rank            = hipcub::Max()(thread_exclusive_suffix_rank  , warp_exclusive_state.x);
        thread_exclusive_segment_index          = hipcub::Sum()(thread_exclusive_segment_index, warp_exclusive_state.y);

        thread_suffix_rank[0]                   = hipcub::Max()(thread_suffix_rank[0], thread_exclusive_suffix_rank);
        thread_suffix_rank[1]                   = hipcub::Max()(thread_suffix_rank[1], thread_exclusive_suffix_rank);
        thread_suffix_rank[2]                   = hipcub::Max()(thread_suffix_rank[2], thread_exclusive_suffix_rank);
        thread_suffix_rank[3]                   = hipcub::Max()(thread_suffix_rank[3], thread_exclusive_suffix_rank);

        thread_segment_index[0]                 = hipcub::Sum()(thread_segment_index[0], thread_exclusive_segment_index);
        thread_segment_index[1]                 = hipcub::Sum()(thread_segment_index[1], thread_exclusive_segment_index);
        thread_segment_index[2]                 = hipcub::Sum()(thread_segment_index[2], thread_exclusive_segment_index);
        thread_segment_index[3]                 = hipcub::Sum()(thread_segment_index[3], thread_exclusive_segment_index);

        const uint32_t thread_index             = blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;

        if (thread_exclusive_segment_index != thread_segment_index[0]) { device_offsets_begin[thread_segment_index[0]] = thread_exclusive_suffix_rank; device_offsets_end[thread_segment_index[0]] = thread_index + 0; }
        if (thread_segment_index[0]        != thread_segment_index[1]) { device_offsets_begin[thread_segment_index[1]] = thread_suffix_rank[0];        device_offsets_end[thread_segment_index[1]] = thread_index + 1; }
        if (thread_segment_index[1]        != thread_segment_index[2]) { device_offsets_begin[thread_segment_index[2]] = thread_suffix_rank[1];        device_offsets_end[thread_segment_index[2]] = thread_index + 2; }
        if (thread_segment_index[2]        != thread_segment_index[3]) { device_offsets_begin[thread_segment_index[3]] = thread_suffix_rank[2];        device_offsets_end[thread_segment_index[3]] = thread_index + 3; }

        if (scatter_ranks_directly)
        {
            const uint4 indexes                 = __ldg((uint4 *)(device_SA + thread_index));

            device_ISA[indexes.x]               = thread_suffix_rank[0];
            device_ISA[indexes.y]               = thread_suffix_rank[1];
            device_ISA[indexes.z]               = thread_suffix_rank[2];
            device_ISA[indexes.w]               = thread_suffix_rank[3];
        }
        else
        {
            *(uint4 *)(device_ISA + thread_index) = make_uint4(thread_suffix_rank[0], thread_suffix_rank[1], thread_suffix_rank[2], thread_suffix_rank[3]);
        }
    }
}

template <bool alternate_block_descriptor_statuses, bool scatter_ranks_directly>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_rank_and_segment_suffixes_incremental_kernel(
    const uint32_t *    RESTRICT device_SA,
    const uint32_t *    RESTRICT device_keys,
    uint8_t *           RESTRICT device_heads,
    uint32_t *          RESTRICT device_out_SA,
    uint32_t *          RESTRICT device_out_ISA,
    uint32_t *          RESTRICT device_offsets_begin,
    uint32_t *          RESTRICT device_offsets_end,
    uint4 *             RESTRICT device_descriptors,
    const uint4 *       RESTRICT device_descriptors_copy
)
{
    __shared__ __align__(32) uint4    warp_state1[1 + CUDA_WARP_THREADS];
    __shared__ __align__(32) uint32_t warp_state2[1 + CUDA_WARP_THREADS];

    uchar4      thread_old_heads;
    uint32_t    thread_exclusive_suffix_old_rank;

    uchar4      thread_new_heads;
    uint32_t    thread_exclusive_suffix_new_rank;

    uint32_t    thread_exclusive_segment_index;
    uint32_t    thread_segment_index[4];

    uint32_t    thread_exclusive_suffix_index;
    uint32_t    thread_suffix_index[4];

    {
        const uint32_t block_index  = blockIdx.x * CUDA_BLOCK_THREADS * 4;
        const uint32_t thread_index = block_index + threadIdx.x * 4;

        device_keys += thread_index; device_heads += thread_index;

        uint2 key_a                 = __ldg((uint2 *)(device_keys - 2));
        uint4 key_b                 = __ldg((uint4 *)(device_keys));
        thread_old_heads            = *(uchar4 *)(device_heads);

        thread_new_heads = make_uchar4(
            (key_a.y != key_b.x) ? (uint8_t)1 : (uint8_t)thread_old_heads.x,
            (key_b.x != key_b.y) ? (uint8_t)1 : (uint8_t)thread_old_heads.y,
            (key_b.y != key_b.z) ? (uint8_t)1 : (uint8_t)thread_old_heads.z,
            (key_b.z != key_b.w) ? (uint8_t)1 : (uint8_t)thread_old_heads.w);

        *(uchar4 *)(device_heads) = thread_new_heads;

        thread_exclusive_suffix_old_rank = (thread_old_heads.x != 0) ? (thread_index + 0) : 0;
        thread_exclusive_suffix_old_rank = (thread_old_heads.y != 0) ? (thread_index + 1) : thread_exclusive_suffix_old_rank;
        thread_exclusive_suffix_old_rank = (thread_old_heads.z != 0) ? (thread_index + 2) : thread_exclusive_suffix_old_rank;
        thread_exclusive_suffix_old_rank = (thread_old_heads.w != 0) ? (thread_index + 3) : thread_exclusive_suffix_old_rank;

        thread_exclusive_suffix_new_rank = (thread_new_heads.x != 0) ? (thread_index + 0) : 0;
        thread_exclusive_suffix_new_rank = (thread_new_heads.y != 0) ? (thread_index + 1) : thread_exclusive_suffix_new_rank;
        thread_exclusive_suffix_new_rank = (thread_new_heads.z != 0) ? (thread_index + 2) : thread_exclusive_suffix_new_rank;
        thread_exclusive_suffix_new_rank = (thread_new_heads.w != 0) ? (thread_index + 3) : thread_exclusive_suffix_new_rank;

        thread_segment_index[0] = ((thread_new_heads.x != 0) && (key_a.x == key_a.y) && (device_heads[-1] == 0));
        thread_segment_index[1] = thread_segment_index[0] + ((thread_new_heads.y != 0) && (thread_new_heads.x == 0));
        thread_segment_index[2] = thread_segment_index[1] + ((thread_new_heads.z != 0) && (thread_new_heads.y == 0));
        thread_segment_index[3] = thread_segment_index[2] + ((thread_new_heads.w != 0) && (thread_new_heads.z == 0));
    }

    {
        uint32_t thread_inclusive_suffix_old_rank;
        uint32_t thread_inclusive_suffix_new_rank;
        uint32_t thread_inclusive_segment_index;

        typedef hipcub::WarpScan<uint32_t> WarpScan;

        __shared__ typename WarpScan::TempStorage warp_scan_storage[CUDA_BLOCK_THREADS];

        WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_exclusive_suffix_old_rank, thread_inclusive_suffix_old_rank, thread_exclusive_suffix_old_rank, (uint32_t)0, hipcub::Max());
        WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_exclusive_suffix_new_rank, thread_inclusive_suffix_new_rank, thread_exclusive_suffix_new_rank, (uint32_t)0, hipcub::Max());
        WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_segment_index[3]         , thread_inclusive_segment_index  , thread_exclusive_segment_index  , (uint32_t)0, hipcub::Sum());

        if ((threadIdx.x % CUDA_WARP_THREADS) == (CUDA_WARP_THREADS - 1))
        {
            warp_state1[threadIdx.x / CUDA_WARP_THREADS] = make_uint4(0, thread_inclusive_suffix_old_rank, thread_inclusive_suffix_new_rank, thread_inclusive_segment_index);
        }

        __syncthreads();
    }

    {
        if (threadIdx.x < CUDA_WARP_THREADS)
        {
            uint32_t            block_exclusive_suffix_new_rank = 0;
            uint32_t            block_exclusive_segment_index   = 0;

            uint32_t            warp_inclusive_suffix_old_rank;
            uint32_t            warp_inclusive_suffix_new_rank;
            uint32_t            warp_inclusive_segment_index;

            {
                typedef hipcub::WarpScan<uint32_t> WarpScan;

                __shared__ typename WarpScan::TempStorage warp_scan_storage;

                uint4 warp_inclusive_state = warp_state1[threadIdx.x];

                WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state.y, warp_inclusive_suffix_old_rank, hipcub::Max());
                WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state.z, warp_inclusive_suffix_new_rank, hipcub::Max());
                WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state.w, warp_inclusive_segment_index  , hipcub::Sum());
            }

            {
                const uint32_t descriptor_status_aggregate_not_ready        = alternate_block_descriptor_statuses ? 4 : 0;
                const uint32_t descriptor_status_partial_aggregate_ready    = alternate_block_descriptor_statuses ? 3 : 1;
                const uint32_t descriptor_status_full_aggregate_ready       = scatter_ranks_directly ? (alternate_block_descriptor_statuses ? 0 : 4) : 2;

                if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                {
                    hipcub::ThreadStore<hipcub::STORE_CG>(device_descriptors + blockIdx.x, make_uint4(descriptor_status_partial_aggregate_ready, 0, warp_inclusive_suffix_new_rank, warp_inclusive_segment_index));
                }

                {
                    uint4 * RESTRICT descriptors_lookback = device_descriptors + blockIdx.x + threadIdx.x;

                    int32_t full_aggregate_lane, delay = 8;
                    do
                    {
                        descriptors_lookback -= CUDA_WARP_THREADS;

                        uint4 block_descriptor;
                        do
                        {
                            libcubwt_delay_or_prevent_hoisting(delay <<= 1);

                            block_descriptor = hipcub::ThreadLoad<hipcub::LOAD_CG>(descriptors_lookback);
                        } while (__any_sync((uint32_t)-1, block_descriptor.x == descriptor_status_aggregate_not_ready));

                        delay = 0;

                        {
                            full_aggregate_lane     = 31 - __clz((int32_t)__ballot_sync((uint32_t)-1, block_descriptor.x != descriptor_status_partial_aggregate_ready));
                            block_descriptor.z      = (((int32_t)threadIdx.x) >= full_aggregate_lane) ? block_descriptor.z : 0;
                            block_descriptor.w      = (((int32_t)threadIdx.x) >= full_aggregate_lane) ? block_descriptor.w : 0;
                        }

                        {
                            block_exclusive_suffix_new_rank     = hipcub::Max()(block_exclusive_suffix_new_rank , libcubwt_warp_reduce_max(block_descriptor.z));
                            block_exclusive_segment_index       = hipcub::Sum()(block_exclusive_segment_index   , libcubwt_warp_reduce_sum(block_descriptor.w));
                        }

                    } while (full_aggregate_lane == -1);

                    warp_inclusive_suffix_new_rank  = hipcub::Max()(warp_inclusive_suffix_new_rank, block_exclusive_suffix_new_rank);
                    warp_inclusive_segment_index    = hipcub::Sum()(warp_inclusive_segment_index  , block_exclusive_segment_index  );
                }

                if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                {
                    hipcub::ThreadStore<hipcub::STORE_CG>(device_descriptors + blockIdx.x, make_uint4(descriptor_status_full_aggregate_ready, 0, warp_inclusive_suffix_new_rank, warp_inclusive_segment_index));
                }
            }

            {
                uint32_t block_exclusive_suffix_old_rank    = __ldg((uint32_t *)(device_descriptors_copy + blockIdx.x - 1) + 2);
                warp_inclusive_suffix_old_rank              = hipcub::Max()(warp_inclusive_suffix_old_rank, block_exclusive_suffix_old_rank);

                if (threadIdx.x == 0)
                {
                    warp_state1[0] = make_uint4(0, block_exclusive_suffix_old_rank, block_exclusive_suffix_new_rank, block_exclusive_segment_index);
                }

                warp_state1[1 + threadIdx.x] = make_uint4(0, warp_inclusive_suffix_old_rank, warp_inclusive_suffix_new_rank, warp_inclusive_segment_index);
            }
        }

        __syncthreads();
    }

    {
        uint32_t thread_suffix_old_rank[4];
        uint32_t thread_suffix_new_rank[4];

        uint4 warp_exclusive_state              = warp_state1[threadIdx.x / CUDA_WARP_THREADS];
        
        thread_exclusive_suffix_old_rank        = hipcub::Max()(thread_exclusive_suffix_old_rank, warp_exclusive_state.y);
        thread_exclusive_suffix_new_rank        = hipcub::Max()(thread_exclusive_suffix_new_rank, warp_exclusive_state.z);
        thread_exclusive_segment_index          = hipcub::Sum()(thread_exclusive_segment_index  , warp_exclusive_state.w);

        const uint32_t thread_index             = blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;

        thread_suffix_old_rank[0]               = (thread_old_heads.x != 0) ? (thread_index + 0) : thread_exclusive_suffix_old_rank;
        thread_suffix_old_rank[1]               = (thread_old_heads.y != 0) ? (thread_index + 1) : thread_suffix_old_rank[0];
        thread_suffix_old_rank[2]               = (thread_old_heads.z != 0) ? (thread_index + 2) : thread_suffix_old_rank[1];
        thread_suffix_old_rank[3]               = (thread_old_heads.w != 0) ? (thread_index + 3) : thread_suffix_old_rank[2];

        thread_suffix_new_rank[0]               = (thread_new_heads.x != 0) ? (thread_index + 0) : thread_exclusive_suffix_new_rank;
        thread_suffix_new_rank[1]               = (thread_new_heads.y != 0) ? (thread_index + 1) : thread_suffix_new_rank[0];
        thread_suffix_new_rank[2]               = (thread_new_heads.z != 0) ? (thread_index + 2) : thread_suffix_new_rank[1];
        thread_suffix_new_rank[3]               = (thread_new_heads.w != 0) ? (thread_index + 3) : thread_suffix_new_rank[2];

        thread_segment_index[0]                 = hipcub::Sum()(thread_segment_index[0], thread_exclusive_segment_index);
        thread_segment_index[1]                 = hipcub::Sum()(thread_segment_index[1], thread_exclusive_segment_index);
        thread_segment_index[2]                 = hipcub::Sum()(thread_segment_index[2], thread_exclusive_segment_index);
        thread_segment_index[3]                 = hipcub::Sum()(thread_segment_index[3], thread_exclusive_segment_index);

        if (thread_exclusive_segment_index != thread_segment_index[0]) { device_offsets_begin[thread_segment_index[0]] = thread_exclusive_suffix_new_rank; device_offsets_end[thread_segment_index[0]] = thread_index + 0; }
        if (thread_segment_index[0]        != thread_segment_index[1]) { device_offsets_begin[thread_segment_index[1]] = thread_suffix_new_rank[0];        device_offsets_end[thread_segment_index[1]] = thread_index + 1; }
        if (thread_segment_index[1]        != thread_segment_index[2]) { device_offsets_begin[thread_segment_index[2]] = thread_suffix_new_rank[1];        device_offsets_end[thread_segment_index[2]] = thread_index + 2; }
        if (thread_segment_index[2]        != thread_segment_index[3]) { device_offsets_begin[thread_segment_index[3]] = thread_suffix_new_rank[2];        device_offsets_end[thread_segment_index[3]] = thread_index + 3; }

        if (scatter_ranks_directly)
        {
            const uint4    indexes              = __ldg((uint4 *)(device_SA + thread_index));

            if (thread_suffix_old_rank[0] != thread_suffix_new_rank[0])  { device_out_ISA[indexes.x] = thread_suffix_new_rank[0]; }
            if (thread_suffix_old_rank[1] != thread_suffix_new_rank[1])  { device_out_ISA[indexes.y] = thread_suffix_new_rank[1]; }
            if (thread_suffix_old_rank[2] != thread_suffix_new_rank[2])  { device_out_ISA[indexes.z] = thread_suffix_new_rank[2]; }
            if (thread_suffix_old_rank[3] != thread_suffix_new_rank[3])  { device_out_ISA[indexes.w] = thread_suffix_new_rank[3]; }
        }
        else
        {
            thread_suffix_index[0]              = (thread_suffix_old_rank[0] != thread_suffix_new_rank[0]);
            thread_suffix_index[1]              = thread_suffix_index[0] + (thread_suffix_old_rank[1] != thread_suffix_new_rank[1]);
            thread_suffix_index[2]              = thread_suffix_index[1] + (thread_suffix_old_rank[2] != thread_suffix_new_rank[2]);
            thread_suffix_index[3]              = thread_suffix_index[2] + (thread_suffix_old_rank[3] != thread_suffix_new_rank[3]);
        }
    }

    if (!scatter_ranks_directly)
    {
        {
            uint32_t thread_inclusive_suffix_index;

            typedef hipcub::WarpScan<uint32_t> WarpScan;

            __shared__ typename WarpScan::TempStorage warp_scan_storage[CUDA_WARP_THREADS];

            WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_suffix_index[3], thread_inclusive_suffix_index, thread_exclusive_suffix_index, (uint32_t)0, hipcub::Sum());

            if ((threadIdx.x % CUDA_WARP_THREADS) == (CUDA_WARP_THREADS - 1))
            {
                warp_state2[threadIdx.x / CUDA_WARP_THREADS] = thread_inclusive_suffix_index;
            }

            __syncthreads();
        }

        {
            if (threadIdx.x < CUDA_WARP_THREADS)
            {
                uint32_t            block_exclusive_suffix_index = 0;
                uint32_t            warp_inclusive_suffix_index;

                {
                    typedef hipcub::WarpScan<uint32_t> WarpScan;

                    __shared__ typename WarpScan::TempStorage warp_scan_storage;

                    uint32_t warp_inclusive_state = warp_state2[threadIdx.x];

                    WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state, warp_inclusive_suffix_index, hipcub::Sum());
                }

                {
                    const uint32_t descriptor_status_aggregate_not_ready        = alternate_block_descriptor_statuses ? 2 : 2;
                    const uint32_t descriptor_status_partial_aggregate_ready    = alternate_block_descriptor_statuses ? 1 : 3;
                    const uint32_t descriptor_status_full_aggregate_ready       = alternate_block_descriptor_statuses ? 0 : 4;

                    if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                    {
                        hipcub::ThreadStore<hipcub::STORE_CG>((uint2 *)(device_descriptors + blockIdx.x), make_uint2(descriptor_status_partial_aggregate_ready, warp_inclusive_suffix_index));
                    }

                    {
                        uint4 * RESTRICT descriptors_lookback = device_descriptors + blockIdx.x + threadIdx.x;

                        int32_t full_aggregate_lane, delay = 8;
                        do
                        {
                            descriptors_lookback -= CUDA_WARP_THREADS;

                            uint2 block_descriptor;
                            do
                            {
                                libcubwt_delay_or_prevent_hoisting(delay <<= 1);

                                block_descriptor = hipcub::ThreadLoad<hipcub::LOAD_CG>((uint2 *)descriptors_lookback);
                            } while (__any_sync((uint32_t)-1, alternate_block_descriptor_statuses 
                                ? ((int32_t )block_descriptor.x >= (int32_t )descriptor_status_aggregate_not_ready)
                                : ((uint32_t)block_descriptor.x <= (uint32_t)descriptor_status_aggregate_not_ready)));

                            delay = 0;

                            {
                                full_aggregate_lane = 31 - __clz((int32_t)__ballot_sync((uint32_t)-1, block_descriptor.x != descriptor_status_partial_aggregate_ready));
                                block_descriptor.y  = (((int32_t)threadIdx.x) >= full_aggregate_lane) ? block_descriptor.y : 0;
                            }

                            {
                                block_exclusive_suffix_index = hipcub::Sum()(block_exclusive_suffix_index, libcubwt_warp_reduce_sum(block_descriptor.y));
                            }

                        } while (full_aggregate_lane == -1);

                        warp_inclusive_suffix_index = hipcub::Sum()(warp_inclusive_suffix_index, block_exclusive_suffix_index);
                    }

                    if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                    {
                        hipcub::ThreadStore<hipcub::STORE_CG>((uint2 *)(device_descriptors + blockIdx.x), make_uint2(descriptor_status_full_aggregate_ready, warp_inclusive_suffix_index));
                    }
                }

                {
                    if (threadIdx.x == 0)
                    {
                        warp_state2[0] = block_exclusive_suffix_index;
                    }

                    warp_state2[1 + threadIdx.x] = warp_inclusive_suffix_index;
                }
            }

            __syncthreads();
        }

        {
            if (thread_suffix_index[3] > 0)
            {
                uint32_t thread_suffix_new_rank[4];

                uint32_t warp_exclusive_state           = warp_state2[threadIdx.x / CUDA_WARP_THREADS];
                thread_exclusive_suffix_index           = hipcub::Sum()(thread_exclusive_suffix_index, warp_exclusive_state);

                thread_suffix_index[0]                  = hipcub::Sum()(thread_suffix_index[0], thread_exclusive_suffix_index);
                thread_suffix_index[1]                  = hipcub::Sum()(thread_suffix_index[1], thread_exclusive_suffix_index);
                thread_suffix_index[2]                  = hipcub::Sum()(thread_suffix_index[2], thread_exclusive_suffix_index);
                thread_suffix_index[3]                  = hipcub::Sum()(thread_suffix_index[3], thread_exclusive_suffix_index);

                const uint32_t thread_index             = blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;
                const uint4    indexes                  = __ldg((uint4 *)(device_SA + thread_index));

                thread_suffix_new_rank[0]               = (thread_new_heads.x != 0) ? (thread_index + 0) : thread_exclusive_suffix_new_rank;
                thread_suffix_new_rank[1]               = (thread_new_heads.y != 0) ? (thread_index + 1) : thread_suffix_new_rank[0];
                thread_suffix_new_rank[2]               = (thread_new_heads.z != 0) ? (thread_index + 2) : thread_suffix_new_rank[1];
                thread_suffix_new_rank[3]               = (thread_new_heads.w != 0) ? (thread_index + 3) : thread_suffix_new_rank[2];

                if (thread_exclusive_suffix_index != thread_suffix_index[0])  { device_out_SA[thread_suffix_index[0]] = indexes.x; device_out_ISA[thread_suffix_index[0]] = thread_suffix_new_rank[0]; }
                if (thread_suffix_index[0]        != thread_suffix_index[1])  { device_out_SA[thread_suffix_index[1]] = indexes.y; device_out_ISA[thread_suffix_index[1]] = thread_suffix_new_rank[1]; }
                if (thread_suffix_index[1]        != thread_suffix_index[2])  { device_out_SA[thread_suffix_index[2]] = indexes.z; device_out_ISA[thread_suffix_index[2]] = thread_suffix_new_rank[2]; }
                if (thread_suffix_index[2]        != thread_suffix_index[3])  { device_out_SA[thread_suffix_index[3]] = indexes.w; device_out_ISA[thread_suffix_index[3]] = thread_suffix_new_rank[3]; }
            }
        }
    }
}

static hipError_t libcubwt_rank_and_segment_suffixes(LIBCUBWT_DEVICE_STORAGE * storage, int64_t n, int64_t iteration)
{
    hipError_t status                      = hipSuccess;
    int64_t     n_segmentation_blocks       = 1 + (n / (storage->cuda_block_threads * 4));
    int64_t     n_initialization_blocks     = (n_segmentation_blocks + storage->cuda_block_threads - 1) / storage->cuda_block_threads;
    bool        scatter_ranks_directly      = (n <= ((int64_t)1 << (storage->device_L2_cache_bits - 3)));

    if (iteration == 0)
    {
        libcubwt_rank_and_segment_suffixes_initialization_kernel<<<(uint32_t)n_initialization_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
            storage->device_SA,
            storage->device_keys_temp_keys,
            storage->device_heads,
            storage->device_descriptors_large,
            storage->device_descriptors_small,
            (uint32_t)n);

        if (scatter_ranks_directly)
        {
            libcubwt_rank_and_segment_suffixes_initiatory_kernel<true><<<(uint32_t)n_segmentation_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                storage->device_SA,
                storage->device_keys_temp_keys,
                storage->device_heads,
                storage->device_ISA,
                storage->device_offsets - 1, storage->device_offsets + (n / 2) - 1,
                storage->device_descriptors_large);
        }
        else
        {
            libcubwt_rank_and_segment_suffixes_initiatory_kernel<false><<<(uint32_t)n_segmentation_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                NULL,
                storage->device_keys_temp_keys,
                storage->device_heads,
                storage->device_temp_ISA,
                storage->device_offsets - 1, storage->device_offsets + (n / 2) - 1,
                storage->device_descriptors_large);
        }

        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->host_pinned_storage, &storage->device_descriptors_large[n_segmentation_blocks - 1], sizeof(uint4), hipMemcpyDeviceToHost, storage->hip_stream));
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamSynchronize(storage->hip_stream), status);

        if (status == hipSuccess)
        {
            storage->num_unsorted_segments = ((uint4 *)storage->host_pinned_storage)->w;

            if (!scatter_ranks_directly)
            {
                if ((status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->device_temp_SA, storage->device_SA, n * sizeof(uint32_t), hipMemcpyDeviceToDevice, storage->hip_stream))) == hipSuccess)
                {
                    status = libcubwt_permute_values_uint32(storage, storage->device_temp_SA, storage->device_temp_ISA, storage->device_ISA, n, storage->device_keys, storage->device_temp_keys);
                }
            }
        }
    }
    else
    {
        if ((status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->device_descriptors_copy - 1, storage->device_descriptors_large - 1, n_segmentation_blocks * sizeof(uint4), hipMemcpyDeviceToDevice, storage->hip_stream))) == hipSuccess)
        {
            if (scatter_ranks_directly)
            {
                if ((iteration % 2) == 0)
                {
                    libcubwt_rank_and_segment_suffixes_incremental_kernel<false, true><<<(uint32_t)n_segmentation_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                        storage->device_SA,
                        storage->device_keys,
                        storage->device_heads,
                        NULL, storage->device_ISA,
                        storage->device_offsets - 1, storage->device_offsets + (n / 2) - 1,
                        storage->device_descriptors_large, storage->device_descriptors_copy);
                }
                else
                {
                    libcubwt_rank_and_segment_suffixes_incremental_kernel<true, true><<<(uint32_t)n_segmentation_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                        storage->device_SA,
                        storage->device_keys,
                        storage->device_heads,
                        NULL, storage->device_ISA,
                        storage->device_offsets - 1, storage->device_offsets + (n / 2) - 1,
                        storage->device_descriptors_large, storage->device_descriptors_copy);
                }
            }
            else
            {
                if ((iteration % 2) == 0)
                {
                    libcubwt_rank_and_segment_suffixes_incremental_kernel<false, false><<<(uint32_t)n_segmentation_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                        storage->device_SA,
                        storage->device_keys,
                        storage->device_heads,
                        storage->device_temp_SA - 1, storage->device_temp_ISA - 1,
                        storage->device_offsets - 1, storage->device_offsets + (n / 2) - 1,
                        storage->device_descriptors_large, storage->device_descriptors_copy);
                }
                else
                {
                    libcubwt_rank_and_segment_suffixes_incremental_kernel<true, false><<<(uint32_t)n_segmentation_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                        storage->device_SA,
                        storage->device_keys,
                        storage->device_heads,
                        storage->device_temp_SA - 1, storage->device_temp_ISA - 1,
                        storage->device_offsets - 1, storage->device_offsets + (n / 2) - 1,
                        storage->device_descriptors_large, storage->device_descriptors_copy);
                }
            }

            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->host_pinned_storage, &storage->device_descriptors_large[n_segmentation_blocks - 1], sizeof(uint4), hipMemcpyDeviceToHost, storage->hip_stream));
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamSynchronize(storage->hip_stream), status);

            if (status == hipSuccess)
            {
                storage->num_unsorted_segments = ((uint4 *)storage->host_pinned_storage)->w;

                if (!scatter_ranks_directly)
                {
                    uint32_t num_updated_suffixes = ((uint4 *)storage->host_pinned_storage)->y;

                    if (num_updated_suffixes > 0)
                    {
                        status = libcubwt_scatter_values_uint32(storage, storage->device_temp_SA, storage->device_temp_ISA, storage->device_ISA, num_updated_suffixes, n, storage->device_keys, storage->device_temp_keys);
                    }
                }
            }
        }
    }

    return status;
}

template <bool alternate_block_descriptor_statuses>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_gather_unsorted_suffixes_kernel(
    const uint8_t *     RESTRICT device_heads, 
    const uint32_t *    RESTRICT device_SA,
    uint32_t *          RESTRICT device_out_keys,
    uint32_t *          RESTRICT device_out_SA,
    uint2 *             RESTRICT device_descriptors)
{
    __shared__ __align__(32) uint32_t warp_state[1 + CUDA_WARP_THREADS];

    uint32_t    thread_exclusive_suffix_index;
    uint32_t    thread_suffix_index[4];

    {
        device_heads += blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;

        const uchar4    current_heads   = __ldg((uchar4 *)(device_heads));
        const uint8_t   next_head       = current_heads.w > 0 ? __ldg(device_heads + 4) : 0;

        thread_suffix_index[0]          = (current_heads.x + current_heads.y < 2);
        thread_suffix_index[1]          = thread_suffix_index[0] + (current_heads.y + current_heads.z < 2);
        thread_suffix_index[2]          = thread_suffix_index[1] + (current_heads.z + current_heads.w < 2);
        thread_suffix_index[3]          = thread_suffix_index[2] + (current_heads.w +       next_head < 2);
    }

    {
        uint32_t thread_inclusive_suffix_index;

        typedef hipcub::WarpScan<uint32_t> WarpScan;

        __shared__ typename WarpScan::TempStorage warp_scan_storage[CUDA_WARP_THREADS];

        WarpScan(warp_scan_storage[threadIdx.x / CUDA_WARP_THREADS]).Scan(thread_suffix_index[3], thread_inclusive_suffix_index, thread_exclusive_suffix_index, (uint32_t)0, hipcub::Sum());

        if ((threadIdx.x % CUDA_WARP_THREADS) == (CUDA_WARP_THREADS - 1))
        {
            warp_state[threadIdx.x / CUDA_WARP_THREADS] = thread_inclusive_suffix_index;
        }

        __syncthreads();
    }

    {
        if (threadIdx.x < CUDA_WARP_THREADS)
        {
            uint32_t block_exclusive_suffix_index = 0;
            uint32_t warp_inclusive_suffix_index;

            {
                typedef hipcub::WarpScan<uint32_t> WarpScan;

                __shared__ typename WarpScan::TempStorage warp_scan_storage;

                uint32_t warp_inclusive_state = warp_state[threadIdx.x];

                WarpScan(warp_scan_storage).InclusiveScan(warp_inclusive_state, warp_inclusive_suffix_index, hipcub::Sum());
            }

            {
                const uint32_t descriptor_status_aggregate_not_ready        = alternate_block_descriptor_statuses ? 2 : 0;
                const uint32_t descriptor_status_partial_aggregate_ready    = alternate_block_descriptor_statuses ? 1 : 1;
                const uint32_t descriptor_status_full_aggregate_ready       = alternate_block_descriptor_statuses ? 0 : 2;

                if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                {
                    hipcub::ThreadStore<hipcub::STORE_CG>(device_descriptors + blockIdx.x, make_uint2(descriptor_status_partial_aggregate_ready, warp_inclusive_suffix_index));
                }

                {
                    uint2 * RESTRICT descriptors_lookback = device_descriptors + blockIdx.x + threadIdx.x;

                    int32_t full_aggregate_lane, delay = 8;
                    do
                    {
                        descriptors_lookback -= CUDA_WARP_THREADS;

                        uint2 block_descriptor;
                        do
                        {
                            libcubwt_delay_or_prevent_hoisting(delay <<= 1);

                            block_descriptor = hipcub::ThreadLoad<hipcub::LOAD_CG>(descriptors_lookback);
                        } while (__any_sync((uint32_t)-1, block_descriptor.x == descriptor_status_aggregate_not_ready));

                        delay = 0;

                        {
                            full_aggregate_lane = 31 - __clz((int32_t)__ballot_sync((uint32_t)-1, block_descriptor.x != descriptor_status_partial_aggregate_ready));
                            block_descriptor.y  = (((int32_t)threadIdx.x) >= full_aggregate_lane) ? block_descriptor.y : 0;
                        }

                        {
                            block_exclusive_suffix_index = hipcub::Sum()(block_exclusive_suffix_index, libcubwt_warp_reduce_sum(block_descriptor.y));
                        }

                    } while (full_aggregate_lane == -1);

                    warp_inclusive_suffix_index = hipcub::Sum()(warp_inclusive_suffix_index, block_exclusive_suffix_index);
                }

                if (threadIdx.x == ((CUDA_BLOCK_THREADS / CUDA_WARP_THREADS) - 1))
                {
                    hipcub::ThreadStore<hipcub::STORE_CG>(device_descriptors + blockIdx.x, make_uint2(descriptor_status_full_aggregate_ready, warp_inclusive_suffix_index));
                }
            }

            {
                if (threadIdx.x == 0)
                {
                    warp_state[0] = block_exclusive_suffix_index;
                }

                warp_state[1 + threadIdx.x] = warp_inclusive_suffix_index;
            }
        }

        __syncthreads();
    }

    {
        if (thread_suffix_index[3] > 0)
        {
            uint32_t warp_exclusive_state           = warp_state[threadIdx.x / CUDA_WARP_THREADS];
        
            thread_exclusive_suffix_index           = hipcub::Sum()(thread_exclusive_suffix_index, warp_exclusive_state);

            thread_suffix_index[0]                  = hipcub::Sum()(thread_suffix_index[0], thread_exclusive_suffix_index);
            thread_suffix_index[1]                  = hipcub::Sum()(thread_suffix_index[1], thread_exclusive_suffix_index);
            thread_suffix_index[2]                  = hipcub::Sum()(thread_suffix_index[2], thread_exclusive_suffix_index);
            thread_suffix_index[3]                  = hipcub::Sum()(thread_suffix_index[3], thread_exclusive_suffix_index);

            const uint32_t thread_index             = blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;
            const uint4    indexes                  = __ldg((uint4 *)(device_SA + thread_index));

            if (thread_exclusive_suffix_index != thread_suffix_index[0]) { device_out_keys[thread_suffix_index[0]] = thread_index + 0; device_out_SA[thread_suffix_index[0]] = indexes.x; }
            if (thread_suffix_index[0]        != thread_suffix_index[1]) { device_out_keys[thread_suffix_index[1]] = thread_index + 1; device_out_SA[thread_suffix_index[1]] = indexes.y; }
            if (thread_suffix_index[1]        != thread_suffix_index[2]) { device_out_keys[thread_suffix_index[2]] = thread_index + 2; device_out_SA[thread_suffix_index[2]] = indexes.z; }
            if (thread_suffix_index[2]        != thread_suffix_index[3]) { device_out_keys[thread_suffix_index[3]] = thread_index + 3; device_out_SA[thread_suffix_index[3]] = indexes.w; }
        }
    }
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_update_suffix_sorting_keys_kernel(const uint8_t * RESTRICT device_heads, const uint32_t * RESTRICT device_SA, const uint32_t * RESTRICT device_ISA, uint32_t * RESTRICT device_keys)
{
    const uint32_t  thread_index    = blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;

    device_heads += thread_index;

    const uchar4    current_heads   = __ldg((uchar4 *)(device_heads));
    const uint8_t   next_head       = current_heads.w > 0 ? __ldg(device_heads + 4) : 0;

    if (current_heads.x + current_heads.y + current_heads.z + current_heads.w + next_head < 5)
    {
        device_SA += thread_index; device_keys += thread_index;

        const uint4 current_SA = __ldg((uint4 *)(device_SA));

        ((uint4 *)device_keys)[0] = make_uint4(
            (current_heads.x + current_heads.y < 2) ? __ldg(device_ISA + current_SA.x) : (uint32_t)-1,
            (current_heads.y + current_heads.z < 2) ? __ldg(device_ISA + current_SA.y) : (uint32_t)-2,
            (current_heads.z + current_heads.w < 2) ? __ldg(device_ISA + current_SA.z) : (uint32_t)-3,
            (current_heads.w +       next_head < 2) ? __ldg(device_ISA + current_SA.w) : (uint32_t)-4);
    }
}

static hipError_t libcubwt_update_suffix_sorting_keys(LIBCUBWT_DEVICE_STORAGE * storage, int64_t n, int64_t iteration, int64_t depth)
{
    hipError_t status                  = hipSuccess;
    int64_t     n_ranking_blocks        = (n + storage->cuda_block_threads * 4 - 1) / (storage->cuda_block_threads * 4);
    bool        gather_keys_directly    = (n <= ((int64_t)1 << (storage->device_L2_cache_bits - 2))) || (n > ((int64_t)1 << (storage->device_L2_cache_bits - 2 + 8)));

    if (gather_keys_directly || (storage->num_unsorted_suffixes <= (n / 4)))
    {
        libcubwt_update_suffix_sorting_keys_kernel<<<(uint32_t)n_ranking_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(storage->device_heads, storage->device_SA, storage->device_ISA + depth, storage->device_keys);
    }
    else
    {
        if ((iteration % 2) == 0)
        {
            libcubwt_gather_unsorted_suffixes_kernel<false><<<(uint32_t)n_ranking_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                storage->device_heads,
                storage->device_SA,
                storage->device_temp_keys - 1, storage->device_temp_SA - 1,
                storage->device_descriptors_small);
        }
        else
        {
            libcubwt_gather_unsorted_suffixes_kernel<true><<<(uint32_t)n_ranking_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                storage->device_heads,
                storage->device_SA,
                storage->device_temp_keys - 1, storage->device_temp_SA - 1,
                storage->device_descriptors_small);
        }

        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->host_pinned_storage, &storage->device_descriptors_small[n_ranking_blocks - 1], sizeof(uint2), hipMemcpyDeviceToHost, storage->hip_stream));
        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamSynchronize(storage->hip_stream), status);

        if (status == hipSuccess)
        {
            storage->num_unsorted_suffixes = ((uint2 *)storage->host_pinned_storage)->y;

            if (storage->num_unsorted_suffixes > 0)
            {
                status = libcubwt_gather_scatter_values_uint32(storage, storage->device_temp_SA, storage->device_ISA + depth, storage->device_temp_keys, storage->device_keys, storage->num_unsorted_suffixes, n, storage->device_temp_ISA, storage->device_keys);
            }
        }
    }

    return status;
}

static hipError_t libcubwt_sort_segmented_suffixes_by_rank(LIBCUBWT_DEVICE_STORAGE * storage, int64_t n)
{
    hipcub::DoubleBuffer<uint32_t> d_keys(storage->device_keys, storage->device_temp_keys);
    hipcub::DoubleBuffer<uint32_t> d_values(storage->device_SA, storage->device_temp_SA);

    hipError_t status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceSegmentedSort::SortPairs(
        storage->device_ssort_temp_storage, storage->device_ssort_temp_storage_size,
        d_keys, d_values,
        (int)storage->num_unsorted_suffixes, (int)storage->num_unsorted_segments,
        storage->device_offsets, storage->device_offsets + (n / 2),
        storage->hip_stream));

    if (d_keys.selector) { std::swap(storage->device_keys, storage->device_temp_keys); }
    if (d_values.selector) { std::swap(storage->device_SA, storage->device_temp_SA); }

    return status;
}

template <bool process_auxiliary_indexes>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_prepare_mod12_suffixes_kernel(const uint8_t * RESTRICT device_T, const uint32_t * RESTRICT device_ISA, const uint64_t * RESTRICT device_suffixes, const uint32_t rm, const uint32_t rs)
{
    __shared__ union
    {
        struct
        {
            __align__(32) uint32_t bytes[4 * CUDA_BLOCK_THREADS];
            __align__(32) uint4    ranks[3 * CUDA_BLOCK_THREADS];
        } stage1;

        struct
        {
            __align__(32) uint4 suffixes[4 * CUDA_BLOCK_THREADS];
        } stage2;

    } shared_storage;

    {
        device_T   += blockIdx.x * CUDA_BLOCK_THREADS * 12 + threadIdx.x * 16;
        device_ISA += blockIdx.x * CUDA_BLOCK_THREADS * 8  + threadIdx.x * 4;

        uint4 * RESTRICT thread_bytes = (uint4 *)shared_storage.stage1.bytes + threadIdx.x;
        uint4 * RESTRICT thread_ranks = (uint4 *)shared_storage.stage1.ranks + threadIdx.x;

        if (threadIdx.x < (12 * CUDA_BLOCK_THREADS + 4 + 15) / 16) { thread_bytes[0] = __ldg((uint4 *)device_T); }

        thread_ranks[0] = __ldg((uint4 *)device_ISA); thread_ranks += CUDA_BLOCK_THREADS; device_ISA += CUDA_BLOCK_THREADS * 4;
        thread_ranks[0] = __ldg((uint4 *)device_ISA); thread_ranks += CUDA_BLOCK_THREADS; device_ISA += CUDA_BLOCK_THREADS * 4;
        if (threadIdx.x == 0) { thread_ranks[0] = __ldg((uint4 *)device_ISA); }
    }

    {
        __syncthreads();

        uint32_t bytes0 = shared_storage.stage1.bytes[threadIdx.x * 3 + 0];
        uint32_t bytes1 = shared_storage.stage1.bytes[threadIdx.x * 3 + 1];
        uint32_t bytes2 = shared_storage.stage1.bytes[threadIdx.x * 3 + 2];
        uint32_t bytes3 = shared_storage.stage1.bytes[threadIdx.x * 3 + 3];

        uint4    ranks0 = shared_storage.stage1.ranks[threadIdx.x * 2 + 0];
        uint4    ranks1 = shared_storage.stage1.ranks[threadIdx.x * 2 + 1];
        uint4    ranks2 = shared_storage.stage1.ranks[threadIdx.x * 2 + 2];

        __syncthreads();

        uint32_t v4 = 0, v8 = 0;

        if (process_auxiliary_indexes)
        {
            const uint32_t i4 = blockIdx.x * CUDA_BLOCK_THREADS * 12 + threadIdx.x * 12 + 4 + rm + 1;
            const uint32_t i8 = blockIdx.x * CUDA_BLOCK_THREADS * 12 + threadIdx.x * 12 + 8 + rm + 1;

            if ((i4 & rm) == 0) { v4 = (i4 >> rs) << 24; }
            if ((i8 & rm) == 0) { v8 = (i8 >> rs) << 24; }
        }

        shared_storage.stage2.suffixes[threadIdx.x * 4 + 0] = make_uint4
        (
            ranks0.y, __byte_perm(bytes0, 0, 0x4021),
            ranks0.z | (uint32_t)INT32_MIN, __byte_perm(bytes0, 0, 0x4132)
        );

        shared_storage.stage2.suffixes[threadIdx.x * 4 + 1] = make_uint4
        (
            ranks0.w, (__byte_perm(bytes0, bytes1, 0x0354) & 0xffffffu) | v4,
            ranks1.x | (uint32_t)INT32_MIN, __byte_perm(bytes1, 0, 0x4021)
        );

        shared_storage.stage2.suffixes[threadIdx.x * 4 + 2] = make_uint4
        (
            ranks1.y, __byte_perm(bytes1, bytes2, 0x0243) & 0xffffffu,
            ranks1.z | (uint32_t)INT32_MIN, (__byte_perm(bytes1, bytes2, 0x0354) & 0xffffffu) | v8
        );

        shared_storage.stage2.suffixes[threadIdx.x * 4 + 3] = make_uint4
        (
            ranks1.w, __byte_perm(bytes2, 0, 0x4132),
            ranks2.x | (uint32_t)INT32_MIN, __byte_perm(bytes2, bytes3, 0x0243) & 0xffffffu
        );

        __syncwarp();
    }

    {
        device_suffixes += blockIdx.x * CUDA_BLOCK_THREADS * 8;

        uint4 * RESTRICT thread_src = shared_storage.stage2.suffixes + ((threadIdx.x / CUDA_WARP_THREADS) * CUDA_WARP_THREADS * 4) + (threadIdx.x % CUDA_WARP_THREADS);
        uint4 * RESTRICT thread_dst = (uint4 *)device_suffixes       + ((threadIdx.x / CUDA_WARP_THREADS) * CUDA_WARP_THREADS * 4) + (threadIdx.x % CUDA_WARP_THREADS);

        thread_dst[0] = thread_src[0]; thread_src += CUDA_WARP_THREADS; thread_dst += CUDA_WARP_THREADS;
        thread_dst[0] = thread_src[0]; thread_src += CUDA_WARP_THREADS; thread_dst += CUDA_WARP_THREADS;
        thread_dst[0] = thread_src[0]; thread_src += CUDA_WARP_THREADS; thread_dst += CUDA_WARP_THREADS;
        thread_dst[0] = thread_src[0];
    }
}

template <bool process_auxiliary_indexes>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_prepare_mod0_suffixes_kernel(const uint8_t * RESTRICT device_T, const uint32_t * RESTRICT device_ISA, const uint64_t * RESTRICT device_suffixes_lh, const uint32_t * RESTRICT device_suffixes_hh, const uint32_t rm, const uint32_t rs)
{
    __shared__ __align__(32) uint16_t bytes[3 * CUDA_BLOCK_THREADS + 8];

    {
        device_T += blockIdx.x * CUDA_BLOCK_THREADS * 6 + threadIdx.x * 16;

        uint4 * RESTRICT thread_bytes = (uint4 *)bytes + threadIdx.x;

        if (threadIdx.x <= (6 * CUDA_BLOCK_THREADS) / 16) { thread_bytes[0] = __ldg((uint4 *)(device_T - 16)); }
    }

    {
        device_ISA         += blockIdx.x * CUDA_BLOCK_THREADS * 4 + threadIdx.x * 4;
        device_suffixes_lh += blockIdx.x * CUDA_BLOCK_THREADS * 2 + threadIdx.x * 2;
        device_suffixes_hh += blockIdx.x * CUDA_BLOCK_THREADS * 2 + threadIdx.x * 2;

        __syncthreads();

        uint32_t bytes0 = bytes[threadIdx.x * 3 + 7 ];
        uint32_t bytes1 = bytes[threadIdx.x * 3 + 8 ];
        uint32_t bytes2 = bytes[threadIdx.x * 3 + 9 ];
        uint32_t bytes3 = bytes[threadIdx.x * 3 + 10];
        uint4    ranks  = __ldg((uint4 *)(device_ISA));

        uint32_t v0 = 0;

        if (process_auxiliary_indexes)
        {
            const uint32_t i0 = blockIdx.x * CUDA_BLOCK_THREADS * 6 + threadIdx.x * 6 + 0 + rm + 1;

            if ((i0 & rm) == 0) { v0 = (i0 >> rs) << 24; }
        }
        else if ((blockIdx.x | threadIdx.x) == 0)
        {
            v0 = 1u << 24;
        }

        *(uint4 *)(device_suffixes_lh) = make_uint4
        (
            ranks.x, __byte_perm(bytes0, bytes1, 0x3154) | v0,
            ranks.z, __byte_perm(bytes2, bytes3, 0x3041)
        );

        *(uint2 *)(device_suffixes_hh) = make_uint2(ranks.y | (uint32_t)INT32_MIN, ranks.w | (uint32_t)INT32_MIN);
    }
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, 1)
static void libcubwt_set_sentinel_suffixes_kernel(uint64_t * RESTRICT device_mod0l_suffixes_end, uint32_t * RESTRICT device_mod0h_suffixes_end,uint64_t * RESTRICT device_mod12_suffixes_end)
{
    uint32_t thread_index = blockIdx.x * CUDA_BLOCK_THREADS + threadIdx.x;

    device_mod0l_suffixes_end += thread_index; 
    device_mod0h_suffixes_end += thread_index;
    device_mod12_suffixes_end += thread_index;

    *(uint2    *)(device_mod0l_suffixes_end) = make_uint2(0x7fffffffu - 12 * CUDA_BLOCK_THREADS + 2 * thread_index + 0, 0x00ffffffu);
    *(uint32_t *)(device_mod0h_suffixes_end) = (uint32_t)(0xffffffffu - 12 * CUDA_BLOCK_THREADS + 2 * thread_index + 0             );
    *(uint2    *)(device_mod12_suffixes_end) = make_uint2(0x7fffffffu - 12 * CUDA_BLOCK_THREADS + 2 * thread_index + 1, 0x00ffffffu);
}

__device__ __forceinline__
bool libcubwt_compare_suffixes_kernel(const uint2 mod0l_suffix, const uint32_t mod0h_suffix, const uint2 mod12_suffix)
{
    uint32_t difference = __byte_perm(mod0l_suffix.y, 0, 0x4401) - __byte_perm(mod12_suffix.y, 0, 0x4401);
    if (difference == 0) { difference = (((int32_t)mod12_suffix.x < 0) ? mod0h_suffix : mod0l_suffix.x) - mod12_suffix.x; }

    return (int32_t)difference <= 0;
}

__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_suffixes_merge_initialization_kernel(
    const uint64_t * RESTRICT device_mod0l_suffixes,
    const uint32_t * RESTRICT device_mod0h_suffixes,
    const uint32_t num_mod0_suffixes,

    const uint64_t * RESTRICT device_mod12_suffixes,
    const uint32_t num_mod12_suffixes,

    uint32_t * RESTRICT device_suffixes_merge_path,
    uint32_t num_merging_blocks)
{
    uint32_t thread_index = blockIdx.x * CUDA_BLOCK_THREADS + threadIdx.x;

    if (thread_index <= num_merging_blocks)
    {
        uint32_t diagonal   = thread_index * CUDA_BLOCK_THREADS * 5;
        uint32_t begin      = (diagonal > num_mod12_suffixes) ? (diagonal - num_mod12_suffixes) : 0;
        uint32_t end        = (diagonal > num_mod0_suffixes ) ? (num_mod0_suffixes            ) : diagonal;

        while (begin < end)
        {
            uint32_t pivot = begin + ((end - begin) >> 1);

            bool predicate = libcubwt_compare_suffixes_kernel(
                __ldg((uint2    *)(device_mod0l_suffixes + pivot)),
                __ldg((uint32_t *)(device_mod0h_suffixes + pivot)),
                __ldg((uint2    *)(device_mod12_suffixes + diagonal - pivot - 1)));

            begin = predicate ? (pivot + 1) : begin;
            end   = predicate ? (end      ) : pivot;
        }

        __syncwarp();

        device_suffixes_merge_path[thread_index] = begin;
    }
}

template <bool process_auxiliary_indexes>
__global__ __launch_bounds__(CUDA_BLOCK_THREADS, CUDA_SM_THREADS / CUDA_BLOCK_THREADS)
static void libcubwt_merge_suffixes_kernel(
    const uint64_t * RESTRICT device_mod0l_suffixes,
    const uint32_t * RESTRICT device_mod0h_suffixes,
    const uint64_t * RESTRICT device_mod12_suffixes,
    const uint32_t * RESTRICT device_suffixes_merge_path,

    uint32_t * RESTRICT device_auxiliary_indexes,
    uint8_t  * RESTRICT device_L)
{
    __shared__ union
    {
        struct
        {
            __align__(32) uint2    suffixes_l[CUDA_BLOCK_THREADS * 5 + 12];
            __align__(32) uint32_t suffixes_h[CUDA_BLOCK_THREADS * 5 + 12];
        } stage1;

        struct
        {
            __align__(32) uint8_t  bwt[CUDA_BLOCK_THREADS * 5];
        } stage2;

    } shared_storage;

    uint32_t num_mod0_suffixes;
    uint32_t num_mod12_suffixes;

    {
        const uint32_t block_mod0_path_begin = (device_suffixes_merge_path + blockIdx.x)[0];
        const uint32_t block_mod0_path_end   = (device_suffixes_merge_path + blockIdx.x)[1];
        
        num_mod0_suffixes   = block_mod0_path_end - block_mod0_path_begin + 6; 
        num_mod12_suffixes  = CUDA_BLOCK_THREADS * 5 + 12 - num_mod0_suffixes;

        device_mod0l_suffixes += block_mod0_path_begin; 
        device_mod0h_suffixes += block_mod0_path_begin;

        device_mod12_suffixes += (blockIdx.x * CUDA_BLOCK_THREADS * 5 - block_mod0_path_begin);
        device_mod12_suffixes -= num_mod0_suffixes;

        #pragma unroll
        for (uint32_t thread_index = threadIdx.x; thread_index < CUDA_BLOCK_THREADS * 5 + 12; thread_index += CUDA_BLOCK_THREADS)
        {
            if (thread_index < num_mod0_suffixes) { shared_storage.stage1.suffixes_h[thread_index] = __ldg(device_mod0h_suffixes + thread_index); }
            shared_storage.stage1.suffixes_l[thread_index] = __ldg((uint2 *)(thread_index < num_mod0_suffixes ? device_mod0l_suffixes : device_mod12_suffixes) + thread_index);
        }

        __syncthreads();
    }

    {
        uint32_t diagonal   = threadIdx.x * 5;
        uint32_t begin      = (diagonal > num_mod12_suffixes) ? (diagonal - num_mod12_suffixes) : 0;
        uint32_t end        = (diagonal > num_mod0_suffixes ) ? (num_mod0_suffixes            ) : diagonal;

        while (begin < end)
        {
            uint32_t pivot = (begin + end) >> 1;

            bool predicate = libcubwt_compare_suffixes_kernel(
                shared_storage.stage1.suffixes_l[pivot],
                shared_storage.stage1.suffixes_h[pivot],
                shared_storage.stage1.suffixes_l[num_mod0_suffixes + diagonal - pivot - 1]);

            begin = predicate ? (pivot + 1) : begin;
            end   = predicate ? (end      ) : pivot;
        }

        __syncwarp();

        uint32_t suffixes[5];

        {
            uint32_t mod0_index     = begin;
            uint32_t mod12_index    = num_mod0_suffixes + diagonal - begin;
            uint2    mod0l_suffix   = shared_storage.stage1.suffixes_l[mod0_index];
            uint32_t mod0h_suffix   = shared_storage.stage1.suffixes_h[mod0_index];
            uint2    mod12_suffix   = shared_storage.stage1.suffixes_l[mod12_index];

            #pragma unroll
            for (uint32_t item = 0; item < 5; ++item)
            {
                bool predicate = libcubwt_compare_suffixes_kernel(mod0l_suffix, mod0h_suffix, mod12_suffix);
           
                suffixes[item] = predicate ? mod0l_suffix.y : mod12_suffix.y;

                if ( predicate) { mod0_index  += 1; mod0l_suffix = shared_storage.stage1.suffixes_l[mod0_index]; mod0h_suffix = shared_storage.stage1.suffixes_h[mod0_index]; }
                if (!predicate) { mod12_index += 1; mod12_suffix = shared_storage.stage1.suffixes_l[mod12_index]; }
            }

            __syncthreads();
        }

        {
            #pragma unroll
            for (uint32_t item = 0; item < 5; ++item)
            {
                if (suffixes[item] >= 0x01000000u)
                {
                    device_auxiliary_indexes[process_auxiliary_indexes ? suffixes[item] >> 24 : 1] = blockIdx.x * CUDA_BLOCK_THREADS * 5 + diagonal + item;
                }

                shared_storage.stage2.bwt[diagonal + item] = (uint8_t)(suffixes[item] >> 16);
            }

            __syncthreads();
        }
    }

    {
        device_L += blockIdx.x * CUDA_BLOCK_THREADS * 5 + threadIdx.x * 16;
        if (threadIdx.x < (CUDA_BLOCK_THREADS * 5 / 16)) { ((uint4 *)device_L)[0] = ((uint4 *)shared_storage.stage2.bwt)[threadIdx.x]; }
    }
}

static hipError_t libcubwt_compute_burrows_wheeler_transform(LIBCUBWT_DEVICE_STORAGE * storage, const uint8_t * T, int64_t input_n, int64_t r, uint32_t * I)
{
    hipError_t status  = hipSuccess;

    int64_t reduced_n   = (input_n   / 3) * 2 + 2;
    int64_t expanded_n  = (reduced_n / 2) * 3 + 0;
    int64_t num_indexes = (input_n + r - 1) / r;

    if ((status = libcubwt_initialize_device_arrays(storage, T, reduced_n, expanded_n, input_n)) == hipSuccess)
    {
        status = libcubwt_sort_suffixes_by_prefix(storage, reduced_n);
    }

    if (status == hipSuccess)
    {
        for (int64_t iteration = 0, depth = 4; true; iteration += 1, depth *= 2)
        {
            if ((status = libcubwt_rank_and_segment_suffixes(storage, reduced_n, iteration)) != hipSuccess)
            {
                break;
            }

            if (storage->num_unsorted_segments == 0)
            {
                break;
            }

            if ((status = libcubwt_update_suffix_sorting_keys(storage, reduced_n, iteration, depth)) != hipSuccess)
            {
                break;
            }

            if ((status = libcubwt_sort_segmented_suffixes_by_rank(storage, reduced_n)) != hipSuccess)
            {
                break;
            }
        }
    }

    if (status == hipSuccess)
    {
        int64_t num_mod0_suffixes  = (input_n / 3) * 1 + ((input_n % 3) != 0);
        int64_t num_mod12_suffixes = (input_n / 3) * 2 + ((input_n % 3) == 2);

        if ((status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(storage->device_temp_ISA, storage->device_ISA, reduced_n * sizeof(uint32_t), hipMemcpyDeviceToDevice, storage->hip_stream))) == hipSuccess)
        {
            hipcub::DoubleBuffer<uint64_t> db_mod12_suffixes(storage->device_keys_temp_keys, storage->device_SA_temp_SA);

            if (status == hipSuccess)
            {
                {
                    int64_t n_preparing_blocks = (num_mod12_suffixes + storage->cuda_block_threads * 8 - 1) / (storage->cuda_block_threads * 8);

                    if (num_indexes > 1)
                    {
                        uint32_t rm = (uint32_t)(r - 1), rs = 0; while (rm >= ((uint32_t)1 << rs)) { rs += 1; }

                        libcubwt_prepare_mod12_suffixes_kernel<true><<<(uint32_t)n_preparing_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                            storage->device_T, storage->device_ISA,
                            db_mod12_suffixes.Current(),
                            rm, rs);
                    }
                    else
                    {
                        libcubwt_prepare_mod12_suffixes_kernel<false><<<(uint32_t)n_preparing_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                            storage->device_T, storage->device_ISA,
                            db_mod12_suffixes.Current(),
                            0, 0);
                    }
                }

                {
                    hipcub::DoubleBuffer<uint32_t> db_index(storage->device_ISA, storage->device_offsets);
                    status = libcubwt_scatter_values_uint64(storage, db_index, db_mod12_suffixes, num_mod12_suffixes, reduced_n, reduced_n - num_mod12_suffixes);
                }
            }

            hipcub::DoubleBuffer<uint32_t> db_mod0h_suffixes(storage->device_ISA, storage->device_offsets);
            hipcub::DoubleBuffer<uint64_t> db_mod0l_suffixes = db_mod12_suffixes.Current() == storage->device_keys_temp_keys
                ? hipcub::DoubleBuffer<uint64_t>((uint64_t *)storage->device_SA, (uint64_t *)storage->device_temp_SA)
                : hipcub::DoubleBuffer<uint64_t>((uint64_t *)storage->device_keys, (uint64_t *)storage->device_temp_keys);

            if (status == hipSuccess)
            {
                {
                    int64_t n_preparing_blocks = (num_mod0_suffixes + storage->cuda_block_threads * 2 - 1) / (storage->cuda_block_threads * 2);

                    if (num_indexes > 1)
                    {
                        uint32_t rm = (uint32_t)(r - 1), rs = 0; while (rm >= ((uint32_t)1 << rs)) { rs += 1; }

                        libcubwt_prepare_mod0_suffixes_kernel<true><<<(uint32_t)n_preparing_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                            storage->device_T, storage->device_temp_ISA,
                            db_mod0l_suffixes.Current(), db_mod0h_suffixes.Current(),
                            rm, rs);
                    }
                    else
                    {
                        libcubwt_prepare_mod0_suffixes_kernel<false><<<(uint32_t)n_preparing_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                            storage->device_T, storage->device_temp_ISA,
                            db_mod0l_suffixes.Current(), db_mod0h_suffixes.Current(),
                            0, 0);
                    }
                }

                if (reduced_n <= (1 << 24))
                {
                    status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
                        storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
                        db_mod0l_suffixes, db_mod0h_suffixes,
                        (uint32_t)num_mod0_suffixes,
                        0, 24,
                        storage->hip_stream));

                    if (status == hipSuccess)
                    {
                        status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
                            storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
                            db_mod0l_suffixes, db_mod0h_suffixes,
                            (uint32_t)num_mod0_suffixes,
                            32, 40,
                            storage->hip_stream));
                    }
                }
                else
                {
                    status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(
                        storage->device_rsort_temp_storage, storage->device_rsort_temp_storage_size,
                        db_mod0l_suffixes, db_mod0h_suffixes,
                        (uint32_t)num_mod0_suffixes,
                        0, 40,
                        storage->hip_stream));
                }
            }

            if (status == hipSuccess)
            {
                int64_t n_merging_blocks = (input_n + storage->cuda_block_threads * 5 - 1) / (storage->cuda_block_threads * 5);

                {
                    libcubwt_set_sentinel_suffixes_kernel<<<6, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                        db_mod0l_suffixes.Current() + num_mod0_suffixes,
                        db_mod0h_suffixes.Current() + num_mod0_suffixes,
                        db_mod12_suffixes.Current() + num_mod12_suffixes);
                }

                {
                    int64_t n_merge_initialization_blocks = 1 + (n_merging_blocks / storage->cuda_block_threads);

                    libcubwt_suffixes_merge_initialization_kernel<<<(uint32_t)n_merge_initialization_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                        db_mod0l_suffixes.Current(), db_mod0h_suffixes.Current(), (uint32_t)(num_mod0_suffixes + 6 * storage->cuda_block_threads),
                        db_mod12_suffixes.Current(), (uint32_t)(num_mod12_suffixes + 6 * storage->cuda_block_threads),
                        (uint32_t *)storage->device_descriptors_large, (uint32_t)n_merging_blocks);
                }

                {
                    if (num_indexes > 1)
                    {
                        libcubwt_merge_suffixes_kernel<true><<<(uint32_t)n_merging_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                            db_mod0l_suffixes.Current(), db_mod0h_suffixes.Current(), db_mod12_suffixes.Current(),
                            (uint32_t *)storage->device_descriptors_large,
                            (uint32_t *)storage->device_descriptors_small - 1,
                            storage->device_T);
                    }
                    else
                    {
                        libcubwt_merge_suffixes_kernel<false><<<(uint32_t)n_merging_blocks, storage->cuda_block_threads, 0, storage->hip_stream>>>(
                            db_mod0l_suffixes.Current(), db_mod0h_suffixes.Current(), db_mod12_suffixes.Current(),
                            (uint32_t *)storage->device_descriptors_large,
                            (uint32_t *)storage->device_descriptors_small - 1,
                            storage->device_T);
                    }
                }
            }

            if (status == hipSuccess)
            {
                uint32_t * buffer = ((sizeof(uint32_t) * num_indexes) <= storage->host_pinned_storage_size) ? (uint32_t *)storage->host_pinned_storage : I;

                status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(buffer, storage->device_descriptors_small, sizeof(uint32_t) * num_indexes, hipMemcpyDeviceToHost, storage->hip_stream), status);

                if ((status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamSynchronize(storage->hip_stream), status)) == hipSuccess)
                {
                    if (I != buffer) { memcpy(I, buffer, sizeof(uint32_t) * num_indexes); }

                    for (int64_t index = 0; index < num_indexes; index += 1) { I[index] += 1; }
                }
            }
        }
    }

    return status;
}

static hipError_t libcubwt_copy_burrows_wheeler_transform(LIBCUBWT_DEVICE_STORAGE * storage, const uint8_t * T, uint8_t * L, int64_t input_n, int64_t index)
{
    hipError_t status = hipSuccess;

    L[0] = T[input_n - 1];

    status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(L + 1, storage->device_T, (size_t)(index - 1), hipMemcpyDeviceToHost, storage->hip_stream), status);
    status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMemcpyAsync(L + index, storage->device_T + index, (size_t)(input_n - index), hipMemcpyDeviceToHost, storage->hip_stream), status);

    status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamSynchronize(storage->hip_stream), status);

    return status;
}

int64_t libcubwt_allocate_device_storage(void ** device_storage, int64_t max_length)
{
    int64_t max_reduced_length  = ((max_length         / 3) * 2 + 2 + 1023) & (-1024);
    int64_t max_expanded_length = ((max_reduced_length / 2) * 3 + 0 + 1023) & (-1024);

    if ((device_storage == NULL) || (max_expanded_length >= INT32_MAX))
    {
        return LIBCUBWT_BAD_PARAMETER;
    }

    *device_storage = NULL;

    LIBCUBWT_DEVICE_STORAGE * storage = (LIBCUBWT_DEVICE_STORAGE *)malloc(sizeof(LIBCUBWT_DEVICE_STORAGE));
    if (storage != NULL)
    {
        memset(storage, 0, sizeof(LIBCUBWT_DEVICE_STORAGE));

        hipError_t status = hipSuccess;

        {
            int32_t cuda_device_ordinal;
            int32_t cuda_device_L2_cache_size;
            int32_t cuda_device_capability;

            libcubwt_cuda_safe_call(__FILE__, __LINE__, hipGetDevice(&cuda_device_ordinal), status);
            libcubwt_cuda_safe_call(__FILE__, __LINE__, hipDeviceGetAttribute(&cuda_device_L2_cache_size, hipDeviceAttributeL2CacheSize, cuda_device_ordinal), status);
            libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::PtxVersion(cuda_device_capability, cuda_device_ordinal), status);

            if (status == hipSuccess)
            {
                storage->device_L2_cache_bits = 0; while (cuda_device_L2_cache_size >>= 1) { storage->device_L2_cache_bits += 1; };

                storage->cuda_block_threads = (cuda_device_capability == 860 || cuda_device_capability == 870 || cuda_device_capability == 890) ? 768 : 512;
            }
        }
               
        if (status == hipSuccess)
        {
            int64_t num_descriptors = ((max_reduced_length / (storage->cuda_block_threads * 4)) + 1024) & (-1024);

            {
                hipcub::DoubleBuffer<uint8_t> uint8_db;
                hipcub::DoubleBuffer<uint32_t> uint32_db;
                hipcub::DoubleBuffer<uint64_t> uint64_db;

                size_t temp_radix_segmented_sort_k32v32 = 0;
                status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceSegmentedSort::SortPairs(NULL, temp_radix_segmented_sort_k32v32, uint32_db, uint32_db, (int)max_reduced_length, (int)max_reduced_length / 2, uint32_db.Current(), uint32_db.Current()), status);

                size_t temp_radix_sort_k32v32 = 0;
                status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(NULL, temp_radix_sort_k32v32, uint32_db, uint32_db, (uint32_t)max_reduced_length), status);

                size_t temp_radix_sort_k64v32 = 0;
                status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(NULL, temp_radix_sort_k64v32, uint64_db, uint32_db, (uint32_t)max_reduced_length), status);

                size_t temp_radix_sort_k32v64 = 0;
                status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipcub::DeviceRadixSort::SortPairs(NULL, temp_radix_sort_k32v64, uint32_db, uint64_db, (uint32_t)max_reduced_length), status);

                storage->device_ssort_temp_storage_size = std::max(temp_radix_segmented_sort_k32v32, (max_reduced_length + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint32_t));
                storage->device_rsort_temp_storage_size = std::max(std::max(temp_radix_sort_k32v32, temp_radix_sort_k64v32), temp_radix_sort_k32v64);

                storage->device_ssort_temp_storage_size = (storage->device_ssort_temp_storage_size + (size_t)1023) & (size_t)(-1024);
                storage->device_rsort_temp_storage_size = (storage->device_rsort_temp_storage_size + (size_t)1023) & (size_t)(-1024);
            }

            if (status == hipSuccess)
            {
                size_t device_storage_size = 0;

                device_storage_size += storage->device_ssort_temp_storage_size;
                device_storage_size += storage->device_rsort_temp_storage_size;

                device_storage_size += (max_expanded_length + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint8_t);

                device_storage_size += (max_reduced_length + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint8_t);
                device_storage_size += (max_reduced_length + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint64_t);
                device_storage_size += (max_reduced_length + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint64_t);
                device_storage_size += (max_reduced_length + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint64_t);

                device_storage_size += (num_descriptors + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint4);
                device_storage_size += (num_descriptors + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint4);
                device_storage_size += (num_descriptors + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint2);

                status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipMalloc((void **)&storage->device_storage, device_storage_size), status);

                if (status == hipSuccess)
                {
                    status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipHostMalloc((void **)&storage->host_pinned_storage, storage->host_pinned_storage_size = 256 * sizeof(uint32_t)), status);
                    status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamCreate(&storage->hip_stream), status);
                }
            }

            if (status == hipSuccess)
            {
                uint8_t * device_alloc              = (uint8_t *)storage->device_storage;

                storage->device_ssort_temp_storage  = (void *)device_alloc; device_alloc += storage->device_ssort_temp_storage_size;
                storage->device_rsort_temp_storage  = (void *)device_alloc; device_alloc += storage->device_rsort_temp_storage_size;

                storage->device_T                   = (uint8_t  *)(void *)device_alloc + CUDA_DEVICE_PADDING; device_alloc += (max_expanded_length + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint8_t);

                storage->device_heads               = (uint8_t  *)(void *)device_alloc + CUDA_DEVICE_PADDING; device_alloc += (max_reduced_length + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint8_t);
                storage->device_SA_temp_SA          = (uint64_t *)(void *)device_alloc + CUDA_DEVICE_PADDING; device_alloc += (max_reduced_length + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint64_t);
                storage->device_keys_temp_keys      = (uint64_t *)(void *)device_alloc + CUDA_DEVICE_PADDING; device_alloc += (max_reduced_length + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint64_t);
                storage->device_offsets_ISA         = (uint64_t *)(void *)device_alloc + CUDA_DEVICE_PADDING; device_alloc += (max_reduced_length + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint64_t);

                storage->device_descriptors_large   = (uint4    *)(void *)device_alloc + CUDA_DEVICE_PADDING; device_alloc += (num_descriptors + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint4);
                storage->device_descriptors_copy    = (uint4    *)(void *)device_alloc + CUDA_DEVICE_PADDING; device_alloc += (num_descriptors + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint4);
                storage->device_descriptors_small   = (uint2    *)(void *)device_alloc + CUDA_DEVICE_PADDING; device_alloc += (num_descriptors + (int64_t)2 * CUDA_DEVICE_PADDING) * sizeof(uint2);

                storage->device_temp_ISA            = (uint32_t *)(void *)storage->device_ssort_temp_storage + CUDA_DEVICE_PADDING;

                storage->device_SA                  = (uint32_t *)(void *)(storage->device_SA_temp_SA       - CUDA_DEVICE_PADDING) + 1 * CUDA_DEVICE_PADDING;
                storage->device_keys                = (uint32_t *)(void *)(storage->device_keys_temp_keys   - CUDA_DEVICE_PADDING) + 1 * CUDA_DEVICE_PADDING;
                storage->device_offsets             = (uint32_t *)(void *)(storage->device_offsets_ISA      - CUDA_DEVICE_PADDING) + 1 * CUDA_DEVICE_PADDING;

                storage->device_temp_SA             = (uint32_t *)(void *)(storage->device_SA_temp_SA       - CUDA_DEVICE_PADDING) + 3 * CUDA_DEVICE_PADDING + max_reduced_length;
                storage->device_temp_keys           = (uint32_t *)(void *)(storage->device_keys_temp_keys   - CUDA_DEVICE_PADDING) + 3 * CUDA_DEVICE_PADDING + max_reduced_length;
                storage->device_ISA                 = (uint32_t *)(void *)(storage->device_offsets_ISA      - CUDA_DEVICE_PADDING) + 3 * CUDA_DEVICE_PADDING + max_reduced_length;

                storage->max_length                 = max_length;

                *device_storage = storage;
                return LIBCUBWT_NO_ERROR;
            }
        }

        libcubwt_free_device_storage(storage);

        return libcubwt_get_error_code(status);
    }

    return LIBCUBWT_NOT_ENOUGH_MEMORY;
}

int64_t libcubwt_free_device_storage(void * device_storage)
{
    hipError_t status = hipSuccess;

    LIBCUBWT_DEVICE_STORAGE * storage = (LIBCUBWT_DEVICE_STORAGE *)device_storage;
    if (storage != NULL)
    {
        if (storage->device_storage != NULL)
        {
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipStreamDestroy(storage->hip_stream), status);
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipHostFree((void *)storage->host_pinned_storage), status);
            status = libcubwt_cuda_safe_call(__FILE__, __LINE__, hipFree((void *)storage->device_storage), status);
        }

        free(storage);
    }

    return status != hipSuccess ? libcubwt_get_error_code(status) : LIBCUBWT_NO_ERROR;
}

int64_t libcubwt_bwt(void * device_storage, const uint8_t * T, uint8_t * L, int64_t n)
{
    LIBCUBWT_DEVICE_STORAGE * storage = (LIBCUBWT_DEVICE_STORAGE *)device_storage;

    if ((storage == NULL) || (T == NULL) || (L == NULL) || (n < 16) || (n > storage->max_length))
    {
        return LIBCUBWT_BAD_PARAMETER;
    }

    hipError_t status; uint32_t index;
    if ((status = libcubwt_compute_burrows_wheeler_transform(storage, T, n, n, &index)) == hipSuccess &&
        (status = libcubwt_copy_burrows_wheeler_transform(storage, T, L, n, index)) == hipSuccess)
    {
        return index;
    }

    return libcubwt_get_error_code(status);
}

int64_t libcubwt_bwt_aux(void * device_storage, const uint8_t * T, uint8_t * L, int64_t n, int64_t r, uint32_t * I)
{
    LIBCUBWT_DEVICE_STORAGE * storage = (LIBCUBWT_DEVICE_STORAGE *)device_storage;

    if ((storage == NULL) || (T == NULL) || (L == NULL) || (n < 16) || (n > storage->max_length) || (r < 4) || ((r & (r - 1)) != 0) || ((n + r - 1) / r > 255) || (I == NULL))
    {
        return LIBCUBWT_BAD_PARAMETER;
    }

    hipError_t status;
    if ((status = libcubwt_compute_burrows_wheeler_transform(storage, T, n, r, I)) == hipSuccess &&
        (status = libcubwt_copy_burrows_wheeler_transform(storage, T, L, n, I[0])) == hipSuccess)
    {
        return LIBCUBWT_NO_ERROR;
    }

    return libcubwt_get_error_code(status);
}
